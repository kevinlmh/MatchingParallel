#include "hip/hip_runtime.h"
/*
* file: RandomTest.c
*
* Testing the probablistic graph matching algorithm
* by rotating a set of artificial points and then calculating
* the similarity score for the edges
*
* Kevin Liu & Reid Delaney
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "graphMatching.cu"
#include "matlib.cu"

#define PI 3.14159265
#define TEST_SIZE	28 

//function returns a random double value in the interval [0, 2PI]
double randomdoubleAngle() {
	//generate random double between 0 and 1
  double r = (double)rand()/(double)RAND_MAX;
  //scale to the 0 to 2PI range
  r *= (2*PI);
  return r;
}

//function returns a random double value in the interval [0,1]
double randomdouble() {
  //generate random double between 0 and 1
  double r = (double)rand() / (double)RAND_MAX;
  return r;
}

//function takes in a set of points, rotates them and then returns the new set
void rotate(Matrix V1, Matrix V2, double centerX, double centerY) {
  double theta = randomdoubleAngle();
  for(int i=0; i < V1.height; i++){
    *(V2.elements + i * V2.width) = cos(theta)*(*(V1.elements + i * V1.width) - centerX) - sin(theta)*(*(V1.elements + i * V1.width + 1) - centerY) + centerX;
    *(V2.elements + i * V2.width + 1) = sin(theta)*(*(V1.elements + i * V1.width) - centerX) + cos(theta)*(*(V1.elements + i * V1.width + 1) - centerY) + centerY;
  }
}

void pointDistort(Matrix V1, Matrix V2, double centerX, double centerY) {
  double distortionX;
  double distortionY;
  for(int i=0; i < V1.height; i++){
    distortionX = (2*randomdouble()-1)/100;
    distortionY = (2*randomdouble()-1)/100;
    *(V2.elements + i * V2.width) = distortionX + *(V1.elements + i * V1.width);
    *(V2.elements + i * V2.width + 1) = distortionY + *(V2.elements + i * V2.width + 1);
  }
}

//create a matrix of distances between nodes
void neighborDistances(Matrix V1, Matrix neighborDist) {
  double distance = 0;
  
  for(int i = 0; i < V1.height; i++) {
    for(int j = 0; j < V1.height; j++) {
      if(i == j)
        *(neighborDist.elements + i * neighborDist.width + j) = 0;
      else {
        distance = sqrt((*(V1.elements+i*V1.width) - *(V1.elements+j*V1.width))*(*(V1.elements+i*V1.width) - *(V1.elements+j*V1.width)) + (*(V1.elements+i*V1.width+1) - *(V1.elements+j*V1.width+1))*(*(V1.elements+i*V1.width+1)- *(V1.elements+j*V1.width+1)));
        *(neighborDist.elements+i*neighborDist.width+j) = distance;
      }
    }
  }

}

int main(int argc, char *argv[]) {
	// initialize random generator with time as seed
  srand(time(NULL));

  int size = atoi(argv[1]);
  
  Matrix V1, V2;
  V1.width = V2.width = 2;
  V1.height = V2.height = size;
  V1.elements = (double*)malloc(size*2*sizeof(double));
  V2.elements = (double*)malloc(size*2*sizeof(double));
  
  for(int i=0; i < size; i++) {
    *(V2.elements + i * V2.width) = *(V1.elements + i * V1.width) = randomdouble();
    *(V2.elements + i * V2.width + 1) = *(V1.elements + i * V1.width + 1) = randomdouble();
  }
  printMatrix(V1);
	saveMatrix(V1, "output/nodes1.txt");
  
  rotate(V1, V2, 0.5, 0.5);
  pointDistort(V2, V2, 0, 0);
  
  printMatrix(V2);
	saveMatrix(V2, "output/nodes2.txt");
  
 	Matrix neighborDist1, neighborDist2;
	neighborDist1.width = neighborDist1.height = size;
	neighborDist2.width = neighborDist2.height = size;
	neighborDist1.elements = (double*)malloc(size*size*sizeof(double));
	neighborDist2.elements = (double*)malloc(size*size*sizeof(double));
	
  neighborDistances(V1, neighborDist1);
  neighborDistances(V2, neighborDist2);
  
  printf("neighbor Distances 1\n");
  printMatrix(neighborDist1);
	saveMatrix(neighborDist1, "output/edges1.txt");

  printf("neighbor distances 2\n");
  printMatrix(neighborDist2);
	saveMatrix(neighborDist2, "output/edges2.txt");
  
	Matrix X, Y, Z;
	X.width = X.height = size;
	Y.width = Y.height = size;
	Z.width = Z.height = size;
	X.elements = (double *)malloc(size*size*sizeof(double));
	Y.elements = (double *)malloc(size*size*sizeof(double));
	Z.elements = (double *)malloc(size*size*sizeof(double));
	
  graphMatching(neighborDist1, neighborDist2, 1, size, X, Z, Y);
  
  printf("X(hard):\n");
  printMatrix(X);
  printf("Z(soft):\n");
  printMatrix(Z);
  printf("Y(debug):\n");
  printMatrix(Y);
	saveMatrix(X, "output/hard.txt");
	saveMatrix(Z, "output/soft.txt");
}
