#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hypergraphMatching.cu"
#include "matlib.cu"

#define BLOCK_SIZE 32

// exp kernel
__global__
void expKernel(Matrix d_D, double sigma) {
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_D.height || col >= d_D.width) return;
	int idx = row * d_D.width + col;
	d_D.elements[idx] = exp(-d_D.elements[idx] * d_D.elements[idx] / sigma);
}

__global__
void calculateD(Matrix d_G1_col, Matrix d_G2t_row, Matrix d_D) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	// if out of boundary return
	if (y >= d_D.height || x >= d_D.width) return;
	
	d_D.elements[y * d_D.width + x] = d_G1_col.elements[y] - d_G2t_row.elements[x];
}

__global__
void expDkernel(Matrix d_G1_col, Matrix d_G2t_row, Matrix d_Y, double sigma) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_Y.height || col >= d_Y.width) return;
	//int idx = row * d_Y.width + col;
	double d = d_G1_col.elements[row] - d_G2t_row.elements[col];
	double e = exp(-d * d / sigma);
	d_Y.elements[row * d_Y.width + col] += e;
}

__global__
void marginalize(Matrix d_G1, Matrix d_G2t, double sigma, Matrix d_Y) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	// if out of boundary return
	if (y >= d_Y.height || x >= d_Y.width) return;

	// create d_G1_col and d_G2_row	
	Matrix d_G1_col, d_G2t_row;
	d_G1_col.height = d_G1.height;
	d_G1_col.width = 1;
	size_t size = d_G1_col.width * d_G1_col.height * sizeof(double);
	hipError_t err = hipMalloc(&d_G1_col.elements, size);
	printf("CUDA malloc d_G1_col: %s\n", hipGetErrorString(err));
	d_G2t_row.height = 1;
	d_G2t_row.width = d_G2t.width;
	size = d_G2t_row.width * d_G2t_row.height * sizeof(double);
	err = hipMalloc(&d_G2t_row.elements, size);
	printf("CUDA malloc d_G2t_row: %s\n", hipGetErrorString(err));

	// calculate D

	// G1(:,i) invoke getCol kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (d_G1.width + dimBlock.x - 1)/dimBlock.x, (d_G1.height + dimBlock.y - 1)/dimBlock.y );
	getColKernel<<<dimGrid, dimBlock>>>(d_G1, d_G1_col, y);
	
	// G2t(j,:) invoke getRow kernel
	dimGrid = dim3( (d_G2t.width + dimBlock.x - 1)/dimBlock.x, (d_G2t.height + dimBlock.y - 1)/dimBlock.y );
	getRowKernel<<<dimGrid, dimBlock>>>(d_G2t, d_G2t_row, x);

	// calculate d_D
	dimGrid = dim3( (d_Y.width + dimBlock.x - 1)/dimBlock.x, (d_Y.height + dimBlock.y - 1)/dimBlock.y ); 
	expDkernel<<<dimGrid, dimBlock>>>(d_G1_col, d_G2t_row, d_Y, sigma);	

	// free memory space
	hipFree(d_G1_col.elements);
	hipFree(d_G2t_row.elements);

}
	
__global__
void marginalize1(Matrix d_G1, Matrix d_G2t, double sigma, Matrix d_Y) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	// if out of boundary return
	if (y >= d_Y.height || x >= d_Y.width) return;
	
	// create d_D
	Matrix d_D;
	d_D.height = d_Y.height;
	d_D.width = d_Y.width;
	size_t size = d_D.width * d_D.height * sizeof(double);
	hipError_t err = hipMalloc(&d_D.elements, size);
	printf("CUDA malloc d_D: %s\n", hipGetErrorString(err));
	
	// create d_G1_col and d_G2_row	
	Matrix d_G1_col, d_G2t_row;
	d_G1_col.height = d_G1.height;
	d_G1_col.width = 1;
	size = d_G1_col.width * d_G1_col.height * sizeof(double);
	err = hipMalloc(&d_G1_col.elements, size);
	printf("CUDA malloc d_G1_col: %s\n", hipGetErrorString(err));
	d_G2t_row.height = 1;
	d_G2t_row.width = d_G2t.width;
	size = d_G2t_row.width * d_G2t_row.height * sizeof(double);
	err = hipMalloc(&d_G2t_row.elements, size);
	printf("CUDA malloc d_G2t_row: %s\n", hipGetErrorString(err));

	// calculate D

	// G1(:,i) invoke getCol kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (d_G1.width + dimBlock.x - 1)/dimBlock.x, (d_G1.height + dimBlock.y - 1)/dimBlock.y );
	getColKernel<<<dimGrid, dimBlock>>>(d_G1, d_G1_col, y);
	
	// G2t(j,:) invoke getRow kernel
	dimGrid = dim3( (d_G2t.width + dimBlock.x - 1)/dimBlock.x, (d_G2t.height + dimBlock.y - 1)/dimBlock.y );
	getRowKernel<<<dimGrid, dimBlock>>>(d_G2t, d_G2t_row, x);

	// calculate d_D
	dimGrid = dim3( (d_D.width + dimBlock.x - 1)/dimBlock.x, (d_D.height + dimBlock.y - 1)/dimBlock.y ); 
	calculateD<<<dimGrid, dimBlock>>>(d_G1_col, d_G2t_row, d_D);	
	
	// exp((-d.*d)./sigma) invoke exp kernel
	expKernel<<<dimGrid, dimBlock>>>(d_D, sigma);
	
	// write to Y
	matAddKernel<<<dimGrid, dimBlock>>>(d_Y, d_D, d_Y);

	// free memory space
	hipFree(d_G1_col.elements);
	hipFree(d_G2t_row.elements);
	hipFree(d_D.elements);

}


__global__
void marginalize0(Matrix d_G1, Matrix d_G2t, double sigma, Matrix d_Y) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	// if out of boundary return
	if (y >= d_Y.height || x >= d_Y.width) return;

	// create d_G1_col and d_G2_row	
	Matrix d_G1_col, d_G2t_row;
	d_G1_col.height = d_G1.height;
	d_G1_col.width = 1;
	size_t size = d_G1_col.width * d_G1_col.height * sizeof(double);
	hipError_t err = hipMalloc(&d_G1_col.elements, size);
	printf("CUDA malloc d_G1_col: %s\n", hipGetErrorString(err));
	d_G2t_row.height = 1;
	d_G2t_row.width = d_G2t.width;
	size = d_G2t_row.width * d_G2t_row.height * sizeof(double);
	err = hipMalloc(&d_G2t_row.elements, size);
	printf("CUDA malloc d_G2t_row: %s\n", hipGetErrorString(err));
	
	
	// calculate D

	// G1(:,i) invoke getCol kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (d_G1.width + dimBlock.x - 1)/dimBlock.x, (d_G1.height + dimBlock.y - 1)/dimBlock.y );
	getColKernel<<<dimGrid, dimBlock>>>(d_G1, d_G1_col, y);
	
	// G2t(j,:) invoke getRow kernel
	dimGrid = dim3( (d_G2t.width + dimBlock.x - 1)/dimBlock.x, (d_G2t.height + dimBlock.y - 1)/dimBlock.y );
	getRowKernel<<<dimGrid, dimBlock>>>(d_G2t, d_G2t_row, x);

	// create d_D, d_D1, d_D2
	Matrix d_D, d_D1, d_D2;
	d_D.height = d_D1.height = d_D2.height = d_Y.height;
	d_D.width = d_D1.width = d_D2.width = d_Y.width;
	size = d_D.width * d_D.height * sizeof(double);
	err = hipMalloc(&d_D.elements, size);
	printf("CUDA malloc d_D: %s\n", hipGetErrorString(err));
	size = d_D1.width * d_D1.height * sizeof(double);
	err = hipMalloc(&d_D1.elements, size);
	printf("CUDA malloc d_D1: %s\n", hipGetErrorString(err));
	size = d_D2.width * d_D2.height * sizeof(double);
	err = hipMalloc(&d_D2.elements, size);
	printf("CUDA malloc d_D2: %s\n", hipGetErrorString(err));


	// repmat(G1(:,i),1,n2) invoke repmat kernel
	dimGrid = dim3( (d_G1_col.width + dimBlock.x - 1)/dimBlock.x, (d_G1_col.height + dimBlock.y - 1)/dimBlock.y );
	repmatKernel<<<dimGrid, dimBlock>>>(d_G1_col, 1, d_G2t.width, d_D1);

	// repmat(G2t(j,:),n1,1) invoke repmat kernel
	dimGrid = dim3( (d_G2t_row.width + dimBlock.x - 1)/dimBlock.x, (d_G2t_row.height + dimBlock.y - 1)/dimBlock.y );
	repmatKernel<<<dimGrid, dimBlock>>>(d_G2t_row, d_G1.height, 1, d_D2);

	// d_D1 - d_D2 invoke matSub kernel
	dimGrid = dim3( (d_D.width + dimBlock.x - 1)/dimBlock.x, (d_D.height + dimBlock.y - 1)/dimBlock.y );
	matSubKernel<<<dimGrid, dimBlock>>>(d_D1, d_D2, d_D);

	// exp((-d.*d)./sigma) invoke exp kernel
	expKernel<<<dimGrid, dimBlock>>>(d_D, sigma);
	
	// write to Y
	matAddKernel<<<dimGrid, dimBlock>>>(d_Y, d_D, d_Y);

	// free memory space
	hipFree(d_G1_col.elements);
	hipFree(d_G2t_row.elements);
	hipFree(d_D.elements);
	hipFree(d_D1.elements);
	hipFree(d_D2.elements);
	
}

void graphMatching(Matrix G1, Matrix G2, double sigma, int numberOfMatches, Matrix X, Matrix Z, Matrix Y) {
	/****************************************************************************************	
	Algorithm due to R. Zass and A. Shashua.,
 	'Probabilistic Graph and Hypergraph Matching.',
 	Computer Vision and Pattern Recognition (CVPR) Anchorage, Alaska, June 2008.

 	G1  				An size1 by size1 symmetric matrix, with the weight of the first graph edges.
 	G2  				An size2 by size2 symmetric matrix, with the weight of the second graph edges.
 	sigma 	 			Kernel parameter for edge-to-edge correlations.
 	numberOfMatches  	number of matches required. 


 	X [Output]  	a size1 by size2 matrix with the hard matching results.
             		The i,j entry is one iff the i-th feature of the first object
             		match the j-th feature of the second object. Zero otherwise.
 
	Z [Output]  	a size1 by size2 matrix with the soft matching results.
             		The i,j entry is the probablity that the i-th feature of the
             		first object match the j-th feature of the second object.
 
	Y [Output]  	Debug information.
	*****************************************************************************************/
	if (isSymmetric(G1) == 0)
		printf("G1 is not symmetric!\n");
	if (isSymmetric(G2) == 0)
		printf("G2 is not symmetric!\n");
	
	// load G1 to device memory
	Matrix d_G1;
	d_G1.width = G1.width;
	d_G1.height = G1.height;
	size_t size = d_G1.width * d_G1.height * sizeof(double);
	hipError_t err = hipMalloc(&d_G1.elements, size);
	printf("CUDA malloc d_G1: %s\n", hipGetErrorString(err));	
	err = hipMemcpy(d_G1.elements, G1.elements, size, hipMemcpyHostToDevice);	
	printf("Copy G1 to device: %s\n", hipGetErrorString(err));
	
	// load G2 to device memory
	Matrix d_G2;
	d_G2.width = G2.height;
	d_G2.height = G2.width;
	size = d_G2.width * d_G2.height * sizeof(double);
	err = hipMalloc(&d_G2.elements, size);
	printf("CUDA malloc d_G2: %s\n", hipGetErrorString(err));	
	err = hipMemcpy(d_G2.elements, G2.elements, size, hipMemcpyHostToDevice);	
	printf("Copy G2 to device: %s\n", hipGetErrorString(err));
	
	// transpose G2	
	// allocate G2t on device memory
	Matrix d_G2t;
	d_G2t.width = G2.height;
	d_G2t.height = G2.width;
	size = d_G2t.width * d_G2t.height * sizeof(double);
	err = hipMalloc(&d_G2t.elements, size);
	printf("CUDA malloc G2t: %s\n", hipGetErrorString(err));	

	// invoke transpose kernel
	//printf("transpose(G2)\n");
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (d_G2.width + dimBlock.x - 1)/dimBlock.x, (d_G2.height + dimBlock.y - 1)/dimBlock.y );
	transposeKernel<<<dimGrid, dimBlock>>>(d_G2, d_G2t);
	err = hipDeviceSynchronize();
	printf("Run transpose kernel: %s\n", hipGetErrorString(err));
	
	// free d_G2
	hipFree(d_G2.elements);
	
	// make Y an all zero matrix
	// allocate Y on device memory
	Matrix d_Y;
	d_Y.height = Y.height;
	d_Y.width = Y.width; 
	size = d_Y.width * d_Y.height * sizeof(double);
	err = hipMalloc(&d_Y.elements, size);
	printf("CUDA malloc d_Y: %s\n", hipGetErrorString(err));
	// invoke zeros kernel
	dimGrid = dim3( (d_Y.width+dimBlock.x-1)/dimBlock.x, (d_Y.height+dimBlock.y-1)/dimBlock.y );
	zerosKernel<<<dimGrid, dimBlock>>>(d_Y);

	marginalize<<<dimGrid, dimBlock>>>(d_G1, d_G2t, sigma, d_Y);
	err = hipDeviceSynchronize();
	//printf("Run marginalize kernel: %s\n", hipGetErrorString(err));

	// read Y from device memory
	size = Y.width * Y.height * sizeof(double);
	err = hipMemcpy(Y.elements, d_Y.elements, size, hipMemcpyDeviceToHost);
	printf("Copy Y off of device: %s\n",hipGetErrorString(err));
	
	// free some device memory
	hipFree(d_G1.elements);
	hipFree(d_G2t.elements);
	hipFree(d_Y.elements);

	// call hypergraphMatching()
	hypergraphMatching(Y, numberOfMatches, X, Z);
}
