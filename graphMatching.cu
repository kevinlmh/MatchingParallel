#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hypergraphMatching.cu"
#include "matlib.cu"

#define BLOCK_SIZE 32

// exp kernel
__global__
void expKernel(Matrix d_D, double sigma) {
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_D.height || col >= d_D.width) return;
	int idx = row * d_D.width + col;
	d_D.elements[idx] = exp(-d_D.elements[idx] * d_D.elements[idx] / sigma);
}

__global__
void marginalize(Matrix d_G1, Matrix d_G2t, double sigma, Matrix d_Y) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	// if out of boundary return
	if (y >= d_Y.height || x >= d_Y.width) return;

	// create d_G1_col and d_G2_row	
	Matrix d_G1_col, d_G2t_row;
	d_G1_col.height = d_G1.height;
	d_G1_col.width = 1;
	d_G1_col.elements = (double*)malloc(d_G1_col.width * d_G1_col.height * sizeof(double));
	d_G2t_row.height = 1;
	d_G2t_row.width = d_G2t.width;
	d_G2t_row.elements = (double*)malloc(d_G2t_row.width * d_G2t_row.height * sizeof(double));
	
	// create d_D, d_D1, d_D2
	Matrix d_D, d_D1, d_D2;
	d_D.height = d_D1.height = d_D2.height = d_Y.height;
	d_D.width = d_D1.width = d_D2.width = d_Y.width;	
	d_D.elements = (double*)malloc(d_D.width * d_D.height * sizeof(double));
	d_D1.elements = (double*)malloc(d_D1.width * d_D1.height * sizeof(double));
	d_D2.elements = (double*)malloc(d_D2.width * d_D2.height * sizeof(double));
	
	// calculate D
	
	// G1(:,i) invoke getCol kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (d_G1.width + dimBlock.x - 1)/dimBlock.x, (d_G1.height + dimBlock.y - 1)/dimBlock.y );
	getColKernel<<<dimGrid, dimBlock>>>(d_G1, d_G1_col, y);
	
	// G2t(j,:) invoke getRow kernel
	dimGrid = dim3( (d_G2t.width + dimBlock.x - 1)/dimBlock.x, (d_G2t.height + dimBlock.y - 1)/dimBlock.y );
	getRowKernel<<<dimGrid, dimBlock>>>(d_G2t, d_G2t_row, x);
	
	// repmat(G1(:,i),1,n2) invoke repmat kernel
	dimGrid = dim3( (d_G1_col.width + dimBlock.x - 1)/dimBlock.x, (d_G1_col.height + dimBlock.y - 1)/dimBlock.y );
	repmatKernel<<<dimGrid, dimBlock>>>(d_G1_col, 1, d_G2t.width, d_D1);

	// repmat(G2t(j,:),n1,1) invoke repmat kernel
	dimGrid = dim3( (d_G2t_row.width + dimBlock.x - 1)/dimBlock.x, (d_G2t_row.height + dimBlock.y - 1)/dimBlock.y );
	repmatKernel<<<dimGrid, dimBlock>>>(d_G2t_row, d_G1.height, 1, d_D2);
	
	// d_D1 - d_D2 invoke matSub kernel
	dimGrid = dim3( (d_D.width + dimBlock.x - 1)/dimBlock.x, (d_D.height + dimBlock.y - 1)/dimBlock.y );
	matSubKernel<<<dimGrid, dimBlock>>>(d_D1, d_D2, d_D);
		
	// exp((-d.*d)./sigma) invoke exp kernel
	expKernel<<<dimGrid, dimBlock>>>(d_D, sigma);
	
	// write to Y
	matAddKernel<<<dimGrid, dimBlock>>>(d_Y, d_D, d_Y);
	//matAtomicAddKernel<<<dimGrid,dimBlock>>>(d_Y, d_D);

	// free memory space
	free(d_G1_col.elements);
	free(d_G2t_row.elements);
	free(d_D.elements);
	free(d_D1.elements);
	free(d_D2.elements);
}

void graphMatching(Matrix G1, Matrix G2, double sigma, int numberOfMatches, Matrix X, Matrix Z, Matrix Y) {
	/****************************************************************************************	
	Algorithm due to R. Zass and A. Shashua.,
 	'Probabilistic Graph and Hypergraph Matching.',
 	Computer Vision and Pattern Recognition (CVPR) Anchorage, Alaska, June 2008.

 	G1  				An size1 by size1 symmetric matrix, with the weight of the first graph edges.
 	G2  				An size2 by size2 symmetric matrix, with the weight of the second graph edges.
 	sigma 	 			Kernel parameter for edge-to-edge correlations.
 	numberOfMatches  	number of matches required. 

 	X [Output]  	a size1 by size2 matrix with the hard matching results.
             		The i,j entry is one iff the i-th feature of the first object
             		match the j-th feature of the second object. Zero otherwise.
 
	Z [Output]  	a size1 by size2 matrix with the soft matching results.
             		The i,j entry is the probablity that the i-th feature of the
             		first object match the j-th feature of the second object.
 
	Y [Output]  	Debug information.
	*****************************************************************************************/
	if (isSymmetric(G1) == 0)
		printf("G1 is not symmetric!\n");
	if (isSymmetric(G2) == 0)
		printf("G2 is not symmetric!\n");
	
	// load G1 to device memory
	Matrix d_G1;
	d_G1.width = G1.width;
	d_G1.height = G1.height;
	size_t size = d_G1.width * d_G1.height * sizeof(double);
	hipError_t err = hipMalloc(&d_G1.elements, size);
	//printf("CUDA malloc d_G1: %s\n", hipGetErrorString(err));	
	err = hipMemcpy(d_G1.elements, G1.elements, size, hipMemcpyHostToDevice);	
	//printf("Copy G1 to device: %s\n", hipGetErrorString(err));
	
	// load G2 to device memory
	Matrix d_G2;
	d_G2.width = G2.height;
	d_G2.height = G2.width;
	size = d_G2.width * d_G2.height * sizeof(double);
	err = hipMalloc(&d_G2.elements, size);
	//printf("CUDA malloc d_G2: %s\n", hipGetErrorString(err));	
	err = hipMemcpy(d_G2.elements, G2.elements, size, hipMemcpyHostToDevice);	
	//printf("Copy G2 to device: %s\n", hipGetErrorString(err));
	
	// transpose G2	
	// allocate G2t on device memory
	Matrix d_G2t;
	d_G2t.width = G2.height;
	d_G2t.height = G2.width;
	size = d_G2t.width * d_G2t.height * sizeof(double);
	err = hipMalloc(&d_G2t.elements, size);
	//printf("CUDA malloc G2t: %s\n", hipGetErrorString(err));	
	
	// invoke transpose kernel
	//printf("transpose(G2)\n");
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (d_G2.width + dimBlock.x - 1)/dimBlock.x, (d_G2.height + dimBlock.y - 1)/dimBlock.y );
	transposeKernel<<<dimGrid, dimBlock>>>(d_G2, d_G2t);
	err = hipDeviceSynchronize();
	//printf("Run transpose kernel: %s\n", hipGetErrorString(err));
	
	// free d_G2
	hipFree(d_G2.elements);
	
	// make Y an all zero matrix
	// allocate Y on device memory
	Matrix d_Y;
	d_Y.height = Y.height;
	d_Y.width = Y.width; 
	size = d_Y.width * d_Y.height * sizeof(double);
	err = hipMalloc(&d_Y.elements, size);
	//printf("CUDA malloc d_Y: %s\n", hipGetErrorString(err));
	// invoke zeros kernel
	dimGrid = dim3( (d_Y.width+dimBlock.x-1)/dimBlock.x, (d_Y.height+dimBlock.y-1)/dimBlock.y );
	zerosKernel<<<dimGrid, dimBlock>>>(d_Y);

	marginalize<<<dimGrid, dimBlock>>>(d_G1, d_G2t, sigma, d_Y);
	err = hipDeviceSynchronize();
	//printf("Run marginalize kernel: %s\n", hipGetErrorString(err));

	// read Y from device memory
	size = Y.width * Y.height * sizeof(double);
	err = hipMemcpy(Y.elements, d_Y.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy Y off of device: %s\n",hipGetErrorString(err));
		
	// free some device memory
	hipFree(d_G1.elements);
	hipFree(d_G2t.elements);
	hipFree(d_Y.elements);

	// call hypergraphMatching()
	hypergraphMatching(Y, numberOfMatches, X, Z);
}

