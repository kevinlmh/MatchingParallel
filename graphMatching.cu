#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
//#include "test_utils.cu"
#include "hypergraphMatching.cu"
#include "k_utils.cu"

#define BLOCK_SIZE 32

/*__device__ double atomicAdd(double* address, double val) {
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,__double_as_longlong(val +
		__longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}*/

// exp kernel
__global__ 
void expKernel(Matrix d_D, double sigma) {
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row * d_D.width + col;
	if(row >= d_D.height || col >= d_D.width) return;
	d_D.elements[idx] = exp(-d_D.elements[idx] * d_D.elements[idx] / sigma);
}

__global__
void marginalize(Matrix d_G1, Matrix d_G2t, double sigma, Matrix d_Y) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	// if out of boundary return
	if (y >= d_G1.height || x >= d_G1.width) return;
	
	// create d_G1_col and d_G2_row	
	Matrix d_G1_col, d_G2t_row;
	d_G1_col.height = d_G1.height;
	d_G1_col.width = 1;
	d_G1_col.elements = (double*)malloc(d_G1_col.width * d_G1_col.height * sizeof(double));
	d_G2t_row.height = 1;
	d_G2t_row.width = d_G1.width;
	d_G2t_row.elements = (double*)malloc(d_G2t_row.width * d_G2t_row.height * sizeof(double));
	
	// create d_D, d_D1, d_D2
	Matrix d_D, d_D1, d_D2;
	d_D.height = d_D1.height = d_D2.height = d_G1.height;
	d_D.width = d_D1.width = d_D2.width = d_G1.width;	
	d_D.elements = (double*)malloc(d_D.width * d_D.height * sizeof(double));
	d_D1.elements = (double*)malloc(d_D1.width * d_D1.height * sizeof(double));
	d_D2.elements = (double*)malloc(d_D2.width * d_D2.height * sizeof(double));
	
	// calculate D
	
	// G1(:,i) invoke getCol kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (d_G1.width + dimBlock.x - 1)/dimBlock.x, (d_G1.height + dimBlock.y - 1)/dimBlock.y );
	getColKernel<<<dimGrid, dimBlock>>>(d_G1, d_G1_col, y);
	/*hipError_t err = hipDeviceSynchronize();
	printf("Run getCol kernel: %s\n", hipGetErrorString(err));*/
	
	// G2t(j,:) invoke getRow kernel
	dimGrid.x = (d_G2t.width + dimBlock.x - 1)/dimBlock.x;
	dimGrid.y = (d_G2t.height + dimBlock.y - 1)/dimBlock.y;
	getRowKernel<<<dimGrid, dimBlock>>>(d_G2t, d_G2t_row, x);
	/*err = hipDeviceSynchronize();
	printf("Run getRow kernel: %s\n", hipGetErrorString(err));*/
	
	// repmat(G1(:,i),1,n2) invoke repmat kernel
	dimGrid.x = (d_G1_col.width + dimBlock.x - 1)/dimBlock.x;
	dimGrid.y = (d_G1_col.height + dimBlock.y - 1)/dimBlock.y;
	repmatKernel<<<dimGrid, dimBlock>>>(d_G1_col, 1, d_G1.width, d_D1);
	/*err = hipDeviceSynchronize();
	printf("Run repmat kernel: %s\n", hipGetErrorString(err));*/

	// repmat(G2t(j,:),n1,1) invoke repmat kernel
	dimGrid.x = (d_G2t_row.width + dimBlock.x - 1)/dimBlock.x;
	dimGrid.y = (d_G2t_row.height + dimBlock.y - 1)/dimBlock.y;
	repmatKernel<<<dimGrid, dimBlock>>>(d_G2t_row, d_G2t.width, 1, d_D2);
	/*err = hipDeviceSynchronize();
	printf("Run repmat kernel: %s\n", hipGetErrorString(err));*/
	
	// d_D1 - d_D2 invoke matSub kernel
	dimGrid.x = (d_D1.width + dimBlock.x - 1)/dimBlock.x;
	dimGrid.y = (d_D1.height + dimBlock.y - 1)/dimBlock.y;
	matSubKernel<<<dimGrid, dimBlock>>>(d_D1, d_D2, d_D);
	/*err = hipDeviceSynchronize();
	printf("Run matSub kernel: %s\n", hipGetErrorString(err));*/
		
	// exp((-d.*d)./sigma) invoke exp kernel
	dimGrid.x = (d_D.width + dimBlock.x - 1)/dimBlock.x;
	dimGrid.y = (d_D.height + dimBlock.y - 1)/dimBlock.y;
	expKernel<<<dimGrid, dimBlock>>>(d_D, sigma);
	/*err = hipDeviceSynchronize();
	printf("Run exp kernel: %s\n", hipGetErrorString(err));*/
	
	// write to Y
	dimGrid.x = (d_D.width + dimBlock.x - 1)/dimBlock.x;
	dimGrid.y = (d_D.height + dimBlock.y - 1)/dimBlock.y;
	matAddKernel<<<dimGrid, dimBlock>>>(d_Y, d_D, d_Y);
	/*err = hipDeviceSynchronize();
	printf("Run matAdd kernel: %s\n", hipGetErrorString(err));*/
	
	// free memory space
	free(d_G1_col.elements);
	free(d_G2t_row.elements);
	free(d_D.elements);
	free(d_D1.elements);
	free(d_D2.elements);
}

void graphMatching(Matrix G1, Matrix G2, double sigma, int numberOfMatches, Matrix X, Matrix Z, Matrix Y) {
	/****************************************************************************************	
	Algorithm due to R. Zass and A. Shashua.,
 	'Probabilistic Graph and Hypergraph Matching.',
 	Computer Vision and Pattern Recognition (CVPR) Anchorage, Alaska, June 2008.

 	G1  				An size1 by size1 symmetric matrix, with the weight of the first graph edges.
 	G2  				An size2 by size2 symmetric matrix, with the weight of the second graph edges.
 	sigma 	 			Kernel parameter for edge-to-edge correlations.
 	numberOfMatches  	number of matches required. 

 	X [Output]  	a size1 by size2 matrix with the hard matching results.
             		The i,j entry is one iff the i-th feature of the first object
             		match the j-th feature of the second object. Zero otherwise.
 
	Z [Output]  	a size1 by size2 matrix with the soft matching results.
             		The i,j entry is the probablity that the i-th feature of the
             		first object match the j-th feature of the second object.
 
	Y [Output]  	Debug information.
	*****************************************************************************************/
	if (isSymmetric(G1) == 0)
		printf("G1 is not symmetric!\n");
	if (isSymmetric(G2) == 0)
		printf("G2 is not symmetric!\n");
	
	// load G1 to device memory
	Matrix d_G1;
	d_G1.width = G1.width;
	d_G1.height = G1.height;
	size_t size = d_G1.width * d_G1.height * sizeof(double);
	hipError_t err = hipMalloc(&d_G1.elements, size);
	printf("CUDA malloc d_G1: %s\n", hipGetErrorString(err));	
	err = hipMemcpy(d_G1.elements, G1.elements, size, hipMemcpyHostToDevice);	
	printf("Copy G1 to device: %s\n", hipGetErrorString(err));
	
	// load G2 to device memory
	Matrix d_G2;
	d_G2.width = G2.height;
	d_G2.height = G2.width;
	size = d_G2.width * d_G2.height * sizeof(double);
	err = hipMalloc(&d_G2.elements, size);
	printf("CUDA malloc d_G2: %s\n", hipGetErrorString(err));	
	err = hipMemcpy(d_G2.elements, G2.elements, size, hipMemcpyHostToDevice);	
	printf("Copy G2 to device: %s\n", hipGetErrorString(err));
	
	// transpose G2	
	// allocate G2t on device memory
	Matrix d_G2t;
	d_G2t.width = G2.height;
	d_G2t.height = G2.width;
	size = d_G2t.width * d_G2t.height * sizeof(double);
	err = hipMalloc(&d_G2t.elements, size);
	printf("CUDA malloc G2t: %s\n", hipGetErrorString(err));	
	
	// invoke transpose kernel
	printf("transpose()\n");
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (d_G2.width + dimBlock.x - 1)/dimBlock.x, (d_G2.height + dimBlock.y - 1)/dimBlock.y );
	transposeKernel<<<dimGrid, dimBlock>>>(d_G2, d_G2t);
	err = hipDeviceSynchronize();
	printf("Run transpose kernel: %s\n", hipGetErrorString(err));

	// make Y an all zero matrix
	// allocate Y on device memory
	Matrix d_Y;
	d_Y.height = Y.height;
	d_Y.width = Y.width; 
	size = d_Y.width * d_Y.height * sizeof(double);
	err = hipMalloc(&d_Y.elements, size);
	printf("CUDA malloc d_Y: %s\n", hipGetErrorString(err));
	// invoke zeros kernel
	dimGrid.x = (d_Y.width+dimBlock.x-1)/dimBlock.x;
	dimGrid.y = (d_Y.height+dimBlock.y-1)/dimBlock.y;
	zerosKernel<<<dimGrid, dimBlock>>>(d_Y);

	dimGrid.x = (G1.width+dimBlock.x-1)/dimBlock.x;
	dimGrid.y = (G1.height+dimBlock.y-1)/dimBlock.y;
	marginalize<<<dimGrid, dimBlock>>>(d_G1, d_G2, sigma, d_Y);
	err = hipDeviceSynchronize();
	printf("Run marginalize kernel: %s\n", hipGetErrorString(err));

	// read Y from device memory
	size = Y.width * Y.height * sizeof(double);
	err = hipMemcpy(Y.elements, d_Y.elements, size, hipMemcpyDeviceToHost);
	printf("Copy Y off of device: %s\n",hipGetErrorString(err));
		
	// free some device memory
	hipFree(d_G1.elements);
	hipFree(d_G2.elements);
	hipFree(d_G2t.elements);
	
	/*
	// allocate d_X and d_Z on device memory
	Matrix d_X, d_Z;
	d_Z.height = d_X.height = Y.height;
	d_Z.width = d_X.width = Y.width; 
	size = d_Y.width * d_Y.height * sizeof(double);
	err = hipMalloc(&d_X.elements, size);
	printf("CUDA malloc d_X: %s\n", hipGetErrorString(err));
	err = hipMalloc(&d_Z.elements, size);
	printf("CUDA malloc d_Z: %s\n", hipGetErrorString(err));
	*/
	
	// call hypergraphMatching()
	hypergraphMatching(Y, numberOfMatches, X, Z);
	
	/*
	// read X and Z from device memory
	err = hipMemcpy(X.elements, d_X.elements, size, hipMemcpyDeviceToHost);
	printf("Copy X off of device: %s\n",hipGetErrorString(err));
	err = hipMemcpy(Z.elements, d_Z.elements, size, hipMemcpyDeviceToHost);
	printf("Copy Z off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_X.elements);
	hipFree(d_Y.elements);
	hipFree(d_Z.elements);
	*/
}




