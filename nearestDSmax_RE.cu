#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "utils.cu"

//#define BLOCK_SIZE 1024
//#define BLOCK_SIZE_DIM2 32
#define EPS 2.2204e-16

/*typedef struct {
  int width;
  int height;
	double* elements;
} Matrix;
*/
void exactTotalSum(Matrix y, Matrix h, double totalSum, double precision, Matrix X){

	Matrix hAlpha;
	hAlpha.width = h.width;
	hAlpha.height = h.height;
	hAlpha.elements = (double*)malloc(hAlpha.width * hAlpha.height * sizeof(double));

// y and h are vectors, totalSum and precision are scalars
// X is the return vector and length is the length of y, h, and X
	double totalSumMinus = totalSum - precision;
	double curAlpha;

	double Min = minOfArray(h.elements, h.height*h.width);

	curAlpha = -Min + EPS;

	double stepAlpha, newAlpha, newSum;
	if(10 > fabs(curAlpha/10))
		stepAlpha = 10;
	else
		stepAlpha = fabs(curAlpha/10);

	for(int j=0; j < 50; j++){

		newAlpha = curAlpha + stepAlpha;
		newSum = 0;

		matPlusScaler(h, newAlpha, hAlpha);
		matDiv(y, hAlpha, X);
		newSum = arraySum(X.elements, X.width*X.height);

		if(newSum > totalSum) {
			curAlpha = newAlpha;
		} else {
			if (newSum < totalSumMinus)
				stepAlpha = stepAlpha / 2;
			else return;
		}

	}

} // end of function

__global__
void unconstrainedKernel(Matrix d_X){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row*d_X.width+col;
	if(row > d_X.height || col > d_X.width) return;
	if(d_X.elements[idx] < EPS)
		d_X.elements[idx] = EPS;
}

void unconstrainedP(Matrix Y, Matrix H, Matrix X){

	matDiv(Y, H, X);
	
// load A to device memory
	Matrix d_X;
	d_X.width = X.width;
	d_X.height = X.height;
	size_t size = X.width * X.height * sizeof(double);
	hipError_t err = hipMalloc(&d_X.elements, size);
	printf("CUDA malloc X: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_X.elements, X.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (X.width + dimBlock.x - 1)/dimBlock.x, (X.height + dimBlock.y - 1)/dimBlock.y );
	zerosKernel<<<dimGrid, dimBlock>>>(d_X);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read A from device memory
	err = hipMemcpy(X.elements, d_X.elements, size, hipMemcpyDeviceToHost);
	printf("Copy X off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_X.elements);

} // end of function

void maxColSumP(Matrix Y, Matrix H, Matrix maxColSum, double precision, Matrix X){

	unconstrainedP(Y, H, X);

	Matrix Xsum;
	Xsum.height = 1;
	Xsum.width = X.width;
	Xsum.elements = (double*) malloc(X.width * sizeof(double));

	for(int i=0; i < X.height; i++){
		Xsum.elements[i] = arraySum(X.elements + i*X.width, X.width);
	}

	Matrix yCol, hCol, Xcol;
	yCol.width = 1;
	hCol.width = 1;
	Xcol.width = 1;
	yCol.height = Y.height;
	hCol.height = H.height;
	Xcol.height = X.height;
	yCol.elements = (double*)malloc(Y.height * sizeof(double));
	hCol.elements = (double*)malloc(H.height * sizeof(double));
	Xcol.elements = (double*)malloc(X.height * sizeof(double));

	for(int i=0; i < Xsum.width; i++) {
		if(Xsum.elements[i] > maxColSum.elements[i]){

//X(:,i) = exactTotalSum (Y(:,i), H(:,i), maxColSum(i), precision);
			getCol(Y, yCol, i);
			getCol(H, hCol, i);

			exactTotalSum(yCol, hCol, maxColSum.elements[i], precision, Xcol);
			
			for(int j=0; j < X.width; j++){
				X.elements[j*X.width + i] = Xcol.elements[j];
			}

		}
	}

	hipFree(yCol.elements);
	hipFree(hCol.elements);
	hipFree(Xcol.elements);
	hipFree(Xsum.elements);

}

// matrix matDiv kernel called by matDiv()
__global__
void HKernel(Matrix d_A, Matrix d_B, Matrix d_C, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row*d_A.width+col;
	if(row > d_A.height || col > d_A.width) return;
	d_Out.elements[idx] = d_A.elements[idx] - (d_B.elements[idx] / (d_C.elements[idx]+EPS));
}

void H(Matrix A, Matrix B, Matrix C, Matrix Out) {
	if (A.width != B.width || A.height != B.height) {
		printf("Input matrices must have the same dimension!\n");
		return;
	}
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix A to device: %s\n", hipGetErrorString(err));
	
	// load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix B to device: %s\n", hipGetErrorString(err));

	// load C to device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	err = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix C to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
	d_Out.width = Out.width; d_Out.height = Out.height;
	size = Out.width * Out.height * sizeof(double);
	hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	HKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	hipFree(d_Out.elements);

}

// matrix lambda kernel called by lambda()
__global__
void lambdaKernel(Matrix d_A, Matrix d_B, Matrix d_C, Matrix d_D, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row*d_A.width+col;
	if(row > d_A.height || col > d_A.width) return;
	d_Out.elements[idx] = d_A.elements[idx] - (d_B.elements[idx] / (d_C.elements[idx]+EPS)) + (d_B.elements[idx] / (d_D.elements[idx]+EPS));
}

void lambda(Matrix A, Matrix B, Matrix C, Matrix D, Matrix Out) {
	if (A.width != B.width || B.width != C.width || A.height != B.height || B.height != C.height){
		printf("Input matrices must have the same dimension!\n");
		return;
	}
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix A to device: %s\n", hipGetErrorString(err));
	
	// load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix B to device: %s\n", hipGetErrorString(err));

	// load C to device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	err = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix C to device: %s\n", hipGetErrorString(err));
	
	// load C to device memory
	Matrix d_D;
	d_D.width = D.width;
	d_D.height = D.height;
	err = hipMalloc(&d_D.elements, size);
	printf("CUDA malloc D: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_D.elements, D.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix D to device: %s\n", hipGetErrorString(err));

	// allocate Out in device memory
	Matrix d_Out;
	d_Out.width = Out.width; d_Out.height = Out.height;
	size = Out.width * Out.height * sizeof(double);
	hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	lambdaKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, d_D, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	hipFree(d_D.elements);
	hipFree(d_Out.elements);

}

// F = (F1 + F2 + F3) / 3;
// matrix lambda kernel called by lambda()
__global__
void FKernel(Matrix d_A, Matrix d_B, Matrix d_C, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row*d_A.width+col;
	if(row > d_A.height || col > d_A.width) return;
	d_Out.elements[idx] = (d_A.elements[idx] + d_B.elements[idx] + d_C.elements[idx]) / 3;
}

void Fun(Matrix A, Matrix B, Matrix C, Matrix Out) {
	if (A.width != B.width || B.width != C.width || A.height != B.height || B.height != C.height){
		printf("Input matrices must have the same dimension!\n");
		return;
	}
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix A to device: %s\n", hipGetErrorString(err));
	
	// load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix B to device: %s\n", hipGetErrorString(err));

	// load C to device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	err = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix C to device: %s\n", hipGetErrorString(err));

	// allocate Out in device memory
	Matrix d_Out;
	d_Out.width = Out.width; d_Out.height = Out.height;
	size = Out.width * Out.height * sizeof(double);
	hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	FKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	hipFree(d_Out.elements);

}

void nearestDSmax_RE(Matrix Y, Matrix maxRowSum, Matrix maxColSum, double totalSum, double maxLoops, double precision, Matrix F){

	zeros(F);
	int m = Y.width;
	int n = Y.height;
	int size = m * n * sizeof(double);

	Matrix lambda1, lambda2, lambda3;
	lambda1.width = lambda2.width = lambda3.width = m;
	lambda1.height = lambda2.height = lambda3.height = n;
	lambda1.elements = (double*)malloc(size);
	lambda2.elements = (double*)malloc(size);
	lambda3.elements = (double*)malloc(size);

	zeros(lambda1);
	zeros(lambda2);
	zeros(lambda3);

	Matrix F1, F2, F3;
	F1.width = F2.width = F3.width = m;
	F1.height = F2.height = F3.height = n;
	F1.elements = (double*)malloc(size);
	F2.elements = (double*)malloc(size);
	F3.elements = (double*)malloc(size);

	double Ysum = matSum(Y);
	Matrix Ydiv;
	Ydiv.width = m;
	Ydiv.height = n;
	Ydiv.elements = (double*)malloc(size);
	matTimesScaler(Y, 1/Ysum, Ydiv);
	matTimesScaler(Ydiv, totalSum, F1);
	matTimesScaler(F1, 1, F2);
	matTimesScaler(F1, 1, F3);

	Matrix H1, H2, H3;
	H1.width = H2.width = H3.width = m;
	H1.height = H2.height = H3.height = n;
	H1.elements = (double*)malloc(size);
	H2.elements = (double*)malloc(size);
	H3.elements = (double*)malloc(size);

	Matrix F1eps, F2eps, F3eps;
	F1eps.width = F2eps.width = F3eps.width = m;
	F1eps.height = F2eps.height = F3eps.height = n;
	F1eps.elements = (double*)malloc(size);
	F2eps.elements = (double*)malloc(size);
	F3eps.elements = (double*)malloc(size);

	Matrix YdivF1eps, YdivF2eps, YdivF3eps;
	YdivF1eps.width = YdivF2eps.width = YdivF3eps.width = m;
	YdivF1eps.height = YdivF2eps.height = YdivF3eps.height = n;
	YdivF1eps.elements = (double*)malloc(size);
	YdivF2eps.elements = (double*)malloc(size);
	YdivF3eps.elements = (double*)malloc(size);

	Matrix negH1t, negH2, negH3;
	negH1t.width = negH2.width = negH3.width = m;
	negH1t.height = negH2.height = negH3.height = n;
	negH1t.elements = (double*)malloc(size);
	negH2.elements = (double*)malloc(size);
	negH3.elements = (double*)malloc(size);

	Matrix H1t, Yt, F1t, X, Yv, Xp;
	H1t.width = Yt.width = F1t.width = X.width = Yv.width = Xp.width = m;
	H1t.height = Yt.height = F1t.height = X.height = Yv.height = Xp.height = n;
	H1t.elements = (double*)malloc(size);
	Yt.elements = (double*)malloc(size);
	F1t.elements = (double*)malloc(size);
	X.elements = (double*)malloc(size);
	Yv.elements = (double*)malloc(size);
	Xp.elements = (double*)malloc(size);

	Matrix Fdiff1, Fdiff2;
	Fdiff1.width = Fdiff2.width = m;
	Fdiff1.height = Fdiff2.height = n;
	Fdiff1.elements = (double*)malloc(size);
	Fdiff2.elements = (double*)malloc(size); 

	Matrix maxRowSumT;
	maxRowSumT.width = m;
	maxRowSumT.height = 1;
	maxRowSumT.elements = (double*)malloc(size/n);

//for t = 1 : maxLoops
	for(int t=0; t < 50; t++){

// Max row sum
	// H1 = lambda1 - (Y ./ (F3+eps));
		H(lambda1, Y, F3, H1);

	//F1 = maxColSumP(Y', -H1', maxRowSum', precision)';
		//-H1'
		transpose(H1, H1t);
		matTimesScaler(H1t, -1, negH1t);
		//Y'
		transpose(Y, Yt);
		//maxRowSum'
		transpose(maxRowSum, maxRowSumT);
		//maxColSumP(Y', -H1', maxRowSum', precision)'
		maxColSumP(Yt, negH1t, maxRowSumT, EPS, F1t);
		//F1
		transpose(F1t, F1);

	// lambda1 = lambda1 - (Y ./ (F3+eps)) + (Y ./ (F1+eps));
		lambda(lambda1, Y, F3, F1, lambda1);

// Max col sum 
	// H2 = lambda2 - (Y ./ (F1+eps));
		H(lambda2, Y, F1, H2);

	// F2 = maxColSumP (Y, -H2, maxColSum, precision);
		matTimesScaler(H2, -1, negH2);
		maxColSumP(Y, negH2, maxColSum, precision, F2);

	// lambda2 = lambda2 - (Y ./ (F1+eps)) + (Y ./ (F2+eps));
		lambda(lambda2, Y, F1, F2, lambda2);

// Total sum
	// H3 = lambda3 - (Y ./ (F2 + eps));
		H(lambda3, Y, F2, H3);

		for(int i = 0; i < m*n; i++){
			Yv.elements[i] = Y.elements[i];
			negH3.elements[i] = H3.elements[i];
		}

		exactTotalSum(Yv, negH3, totalSum, precision, X);

		reshape(X, F3);

	//lambda3 = lambda3 - (Y ./ (F2+eps)) + (Y ./ (F3+eps));
		lambda(lambda3, Y, F2, F3, lambda3);

		matSub(F1, F2, Fdiff1);
		matSub(F1, F3, Fdiff2);
		double fdMax1 = max(maxOfMatrix(Fdiff1), fabs(minOfMatrix(Fdiff1)));
		double fdMax2 = max(maxOfMatrix(Fdiff2), fabs(minOfMatrix(Fdiff2)));

		if(fabs(fdMax1) < precision && fabs(fdMax2) < precision)
			break;

	} // end of t for loop

	
// F = (F1 + F2 + F3) / 3;
	Fun(F1, F2, F3, F);

	free(lambda1.elements);
	free(lambda2.elements);
	free(lambda3.elements);
	free(F1.elements);
	free(F2.elements);
	free(F3.elements);
	free(H1.elements);
	free(H2.elements);
	free(H3.elements);
	free(F1eps.elements);
	free(F2eps.elements);
	free(F3eps.elements);
	free(YdivF1eps.elements);
	free(YdivF2eps.elements);
	free(YdivF3eps.elements);
	free(H1t.elements);
	free(negH1t.elements);
	free(negH2.elements);
	free(negH3.elements);
	free(Yt.elements);
	free(maxRowSumT.elements);
	free(F1t.elements);
	free(X.elements);
	free(Yv.elements);
	free(Xp.elements);
	free(Fdiff1.elements);
	free(Fdiff2.elements);

}
