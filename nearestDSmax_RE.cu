#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "utils.c"

#define BLOCK_SIZE 1024
#define BLOCK_SIZE_DIM2 32
#define EPS 2.2204e-16

typedef struct {
  int width;
  int height;
	double* elements;
} Matrix;

void exactTotalSum(Matrix y, Matrix h, double totalSum, Matrix X){

// y and h are vectors, totalSum and precision are scalars
// X is the return vector and length is the length of y, h, and X
	double totalSumMinus = totalSum - precision;
	double curAlpha;

	double Min = minOfArray(h, length);

	curAlpha = -Min + EPS;

	double stepAlpha, newAlpha, newSum;
	if(10 > fabs(curAlpha/10))
		stepAlpha = 10;
	else
		stepAlpha = fabs(curAlpha/10);

	for(int j=0; j < 50; j++){

		newAlpha = curAlpha + stepAlpha;
		newSum = 0;

		matPlusScaler(h, hAlpha, newAlpha);
		matDiv(y, hAlpha, X);
		newSum = arraySum(X.elements[0]);

		if(newSum > totalSum) {
			curAlpha = newAlpha;
		} else {
			if (newSum < totalSumMinus)
				stepAlpha = stepAlpha / 2;
			else return;
		}

	}

} // end of function

__global__
void unconstrainedKernel(Matrix d_X){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row*d_A.width+col;
	if(row > d_A.height || col > d_A.width) return;
	if(d_X.elements[idx] < EPS)
		d_X.elements[idx] = EPS;
}

void unconstrainedP(Matrix Y, Matrix H, Matrix X){

	matDiv(Y, H, X);
	
// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, X.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	zerosKernel<<<dimGrid, dimBlock>>>(d_A);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read A from device memory
	err = hipMemcpy(X.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);

} // end of function

void maxColSumP(Matrix Y, Matrix H, Matrix maxColSum, double precision, Matrix X){

	unconstrainedP(Y, H, X);

	Matrix Xsum;
	Xum.height = 1;
	Xsum.width = X.width;
	double* Xsum = (double*) malloc(X.width * sizeof(double));

	for(int i=0; i < X.height; i++){
		Xsum[i] = arraySum(X + i*X.width, X.width);
	}

	Matrix yCol, hCol, Xcol;
	yCol.width = 1;
	hCol.width = 1;
	Xcol.wdith = 1;
	yCol.height = Y.height;
	hCol.height = H.height;
	Xcol.height = X.height;
	double* yCol.elements = (double*)malloc(Y.height * sizeof(double));
	double* hCol.elements = (double*)malloc(H.height * sizeof(double));
	double* Xcol.elements = (double*)malloc(X.height * sizeof(double));

	for(int i=0; i < Xsum.width; i++) {
		if(Xsum[i] > maxColSum[i]){

//X(:,i) = exactTotalSum (Y(:,i), H(:,i), maxColSum(i), precision);
			getCol(Y, yCol, i);
			getCol(H, hCol, i);

			exactTotalSum(yCol, hCol, maxColSum[i], precision, Xcol);
			
			for(int j=0; j < x.width; j++){
				X[j*X.width + i] = Xcol[j];
			}

		}
	}

	hipFree(yCol.elements);
	hipFree(hCol.elements);
	hipFree(Xcol.elements);
	hipFree(Xsum.elements);

}

void nearestDSmax_RE(Matrix Y, Matrix maxRowSum, Matrix maxColSum, double totalSum, double precision, double maxLoops, double precision, Matrix F){

	zeros(F);
	int m = Y.width;
	int n = Y.height;
	int size = m * n * sizeof(double);

	Matrix lambda1, lambda2, lambda3;
	lambda1.width = m;
	lambda2.width = m;
	lambda3.width = m;
	lambda1.height = n;
	lambda2.height = n;
	lambda3.height = n;
	double* lambda1.elements = (double*)malloc(size);
	double* lambda2.elements = (double*)malloc(size);
	double* lambda3.elements = (double*)malloc(size);

	zeros(lambda1);
	zeros(lambda2);
	zeros(lambda3);

	Matrix F1, F2, F3;
	F1.width = m;
	F2.width = m;
	F3.width = m;
	F1.height = n;
	F2.height = n;
	F3.height = n;
	double* F1.elements = (double*)malloc(size);
	double* F2.elements = (double*)malloc(size);
	double* F3.elements = (double*)malloc(size);

	double Ysum = matSum(Y);
	Matrix Ydiv;
	Ydiv.width = m;
	Ydiv.height = n;
	double* Ydiv.elements = size;
	matTimesScaler(Y, 1/Ysum, Ydiv);
	matTimesScaler(Ydiv, totalSum, F1);
	matTimesScaler(F1, 1, F2);
	matTimesScaler(F1, 1, F3);

	Matrix H1, H2, H3;
	H1.width = H2.width = H3.width = m;
	H1.height = H2.height = H3.height = n;
	double* H1.elements = (double*)malloc(size);
	double* H2.elements = (double*)malloc(size);
	double* H3.elements = (double*)malloc(size);

	Matrix F1eps, F2eps, F3eps;
	F1eps.width = F2eps.width = F3eps.width = m;
	F1eps.height = F2eps.height = F3eps.height = n;
	double* F1eps.elements = (double*)malloc(size);
	double* F2eps.elements = (double*)malloc(size);
	double* F3eps.elements = (double*)malloc(size);

	Matrix YdivF1eps, YdivF2eps, YdivF3eps;
	YdivF1eps.width = YdivF2eps.width = YdivF3eps.width = m;
	YdivF1eps.height = YdivF2eps.height = YdivF3eps.height = n;
	double* YdivF1eps.elements = (double*)malloc(size);
	double* YdivF2eps.elements = (double*)malloc(size);
	double* YdivF3eps.elements = (double*)malloc(size);

	Matrix negH1t, negH2t, negH3t;
	negH1t.width = negH2t.width = negH3t.width = m;
	negH1t.height = negH2t.height = negH3t.height = n;
	double* negH1t.elements = (double*)malloc(size);
	double* negH2t.elements = (double*)malloc(size);
	double* negH3t.elements = (double*)malloc(size);

	Matrix H1t, Yt, F1t, X, Yv, Xp;
	H1t.width = Yt.width = F1t.width = X.width = Yv.width = Xp.width = m;
	H1t.height = Yt.height = F1t.height = X.height = Yv.height = Xp.height = n;
	double* H1t.elements = (double*)malloc(size);
	double* Yt.elements = (double*)malloc(size);
	double* F1t.elements = (double*)malloc(size);
	double* X.elements = (double*)malloc(size);
	double* Yv.elements = (double*)malloc(size);
	double* Xp.elements = (double*)malloc(size);

	Matrix Fdiff1, Fdiff2;
	Fdiff1.width = Fdiff2.width = m;
	Fdiff1.height = Fdiff2.height = n;
	double* Fdiff1.elements = (double*)malloc(size);
	double* Fdiff2.elements = (double*)malloc(size); 

	Matrix maxRowSumT;
	maxRowSumT.width = m;
	maxRowSumT.height = 1;
	double* maxRowSumT.elements = (double*)malloc(size/n);

//for t = 1 : maxLoops
	for(int t=0; t < 50; t++){

// Max row sum
	// H1 = lambda1 - (Y ./ (F3+eps));
		H(lambda1, Y, F3, H1);

	//F1 = maxColSumP(Y', -H1', maxRowSum', precision)';
		//-H1'
		transpose(H1, H1t);
		matTimesScaler(H1t, -1, negH1t);
		//Y'
		transpose(Y, Yt);
		//maxRowSum'
		transpose(maxRowSum, maxRowSumT);
		//maxColSumP(Y', -H1', maxRowSum', precision)'
		maxColSumP(Yt, negH1t, maxRowSumT, EPS, F1t);
		//F1
		transpose(F1t, F1);

	// lambda1 = lambda1 - (Y ./ (F3+eps)) + (Y ./ (F1+eps));
		lambda(lambda1, Y, F3, F1, lambda1);

// Max col sum 
	// H2 = lambda2 - (Y ./ (F1+eps));
		H(lambda2, Y, F1, H2);

	// F2 = maxColSumP (Y, -H2, maxColSum, precision);
		matTimesScaler(H2, -1, negH2);
		maxColSumP(Y, negH2, maxColSum, precision, F2);

	// lambda2 = lambda2 - (Y ./ (F1+eps)) + (Y ./ (F2+eps));
		lambda(lambda2, Y, F1, F2, lambda2);

// Total sum
	// H3 = lambda3 - (Y ./ (F2 + eps));
		H(lambda3, Y, F2, H3);

		for(int i = 0; i < m*n; i++){
			Yv.elements[i] = Y.elements[i];
			negH3.elements[i] = H3.elements[i];
		}

		exactTotalSum(Yv, negH3, totalSum, precision, X);

		reshape(X, m, n, F3);

	//lambda3 = lambda3 - (Y ./ (F2+eps)) + (Y ./ (F3+eps));
		lambda(lambda3, Y, F2, F3, lambda3);

		matSub(F1, F2, Fdiff1);
		matSub(F1, F3, Fdiff2);
		double fdMax1 = max(maxOfMatrix(Fdiff1), fabs(minOfMatrix(Fdiff1)));
		double fdMax2 = max(maxOfMatrix(Fdiff2), fabs(minOfMatrix(Fdiff2)));

		if(fabs(fdMax1) < precision && fabs(fdMax2) < precision)
			break;

	} // end of t for loop

	Matrix F;
	F.width = F1.width;
	F.height = F1.height;
	double* F.elements = (double*)malloc(size);

// F = (F1 + F2 + F3) / 3;
	F(F1, F2, F3, F);

	hipFree(lambda1);
	hipFree(lambda2);
	hipFree(lambda3);
	hipFree(F1);
	hipFree(F2);
	hipFree(F3);
	hipFree(H1);
	hipFree(H2);
	hipFree(H3);
	hipFree(F1eps);
	hipFree(F2eps);
	hipFree(F3eps);
	hipFree(YdivF1eps);
	hipFree(YdivF2eps);
	hipFree(YdivF3eps);
	hipFree(H1t);
	hipFree(negH1t);
	hipFree(negH2);
	hipFree(negH3);
	hipFree(Yt);
	hipFree(maxRowSumT);
	hipFree(F1t);
	hipFree(X);
	hipFree(Yv);
	hipFree(Xp);
	hipFree(Fdiff1);
	hipFree(Fdiff2);
	hipFree(Fdiv);

}

// matrix matDiv kernel called by matDiv()
__global__
void HKernel(Matrix d_A, Matrix d_B, Matrix d_C, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row*d_A.width+col;
	if(row > d_A.height || col > d_A.width) return;
	d_Out.elements[idx] = d_A.elements[idx] - (d_B.elements[idx] / (d_C.elements[idx]+EPS));
}

void H(Matrix A, Matrix B, Matrix C, Matrix Out) {
	if (A.width != B.width || A.height != B.height) {
		printf("Input matrices must have the same dimension!\n");
		return;
	}
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix A to device: %s\n", hipGetErrorString(err));
	
	// load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix B to device: %s\n", hipGetErrorString(err));

	// load C to device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	err = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix C to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
	d_Out.width = Out.width; d_Out.height = Out.height;
	size = Out.width * Out.height * sizeof(double);
	hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	H1Kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	hipFree(d_Out.elements);

}

// lambda1 = lambda1 - (Y ./ (F3+eps)) + (Y ./ (F1+eps));
		matPlusScaler(F1, EPS, F1eps);
		matDiv(Y, F1eps, YdivF1eps);
		matSub(lambda1, YdivF3eps, lambda1);
		matAdd(lambda1, YdivF1eps, lambda1);

// matrix lambda kernel called by lambda()
__global__
void lambdaKernel(Matrix d_A, Matrix d_B, Matrix d_C, Matrix d_D, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row*d_A.width+col;
	if(row > d_A.height || col > d_A.width) return;
	d_Out.elements[idx] = d_A.elements[idx] - (d_B.elements[idx] / (d_C.elements[idx]+EPS)) + (d_B.elements[idx] / (d_D.elements[idx]+EPS));
}

void lambda(Matrix A, Matrix B, Matrix C, Matrix D, Matrix Out) {
	if (A.width != B.width || B.width != C.width || A.height != B.height || B.height != C.height){
		printf("Input matrices must have the same dimension!\n");
		return;
	}
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix A to device: %s\n", hipGetErrorString(err));
	
	// load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix B to device: %s\n", hipGetErrorString(err));

	// load C to device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	err = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix C to device: %s\n", hipGetErrorString(err));
	
	// load C to device memory
	Matrix d_D;
	d_D.width = D.width;
	d_D.height = D.height;
	err = hipMalloc(&d_D.elements, size);
	printf("CUDA malloc D: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_D.elements, D.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix D to device: %s\n", hipGetErrorString(err));

	// allocate Out in device memory
	Matrix d_Out;
	d_Out.width = Out.width; d_Out.height = Out.height;
	size = Out.width * Out.height * sizeof(double);
	hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	H1Kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, d_D, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	hipFree(d_D.elements);
	hipFree(d_Out.elements);

}

// F = (F1 + F2 + F3) / 3;
// matrix lambda kernel called by lambda()
__global__
void Fkernel(Matrix d_A, Matrix d_B, Matrix d_C, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row*d_A.width+col;
	if(row > d_A.height || col > d_A.width) return;
	d_Out.elements[idx] = (d_A.elements[idx] + d_B.elements[idx] + d_C.elements[idx]) / 3;
}

void F(Matrix A, Matrix B, Matrix C, Matrix Out) {
	if (A.width != B.width || B.width != C.width || A.height != B.height || B.height != C.height){
		printf("Input matrices must have the same dimension!\n");
		return;
	}
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix A to device: %s\n", hipGetErrorString(err));
	
	// load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix B to device: %s\n", hipGetErrorString(err));

	// load C to device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	err = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix C to device: %s\n", hipGetErrorString(err));

	// allocate Out in device memory
	Matrix d_Out;
	d_Out.width = Out.width; d_Out.height = Out.height;
	size = Out.width * Out.height * sizeof(double);
	hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	FKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	hipFree(d_Out.elements);

}
