#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>

#define BLOCK_SIZE 1024
#define BLOCK_SIZE_DIM2 32
// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
/*typedef struct {
  int width;
  int height;
	double* elements;
} Matrix;
*/

__global__
void arraySumKernel(double *elements, int size, double *d_part) {
	// Reduction max, works for any blockDim.x:
	int  thread2;
	double temp;
	__shared__ double sdata[BLOCK_SIZE];
	
	// Load max from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		sdata[threadIdx.x] = elements[idx];
	else
		sdata[threadIdx.x] = DBL_MIN;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE;	// Total number of threads, rounded up to the next power of two
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;

			// Skipping the fictious threads blockDim.x ... blockDim_2-1
			if (thread2 < blockDim.x) {
				// Get the shared value stored by another thread 
				temp = sdata[thread2];
					 sdata[threadIdx.x] += temp;
			}
		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the max to d_max
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

/*int NearestPowerOf2(int n) {
  if (!n) return n;  //(0 == 2^0)
  int x = 1;
  while(x < n) {
      x <<= 1;
  }
  return x;
}*/

double arraySum(double* A, int elements) {
	hipEvent_t start, stop;
	float time;
	// create events and start the timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	// load A to device memory
	double* d_A;
	size_t size = elements * sizeof(double);
	hipError_t err = hipMalloc(&d_A, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load d_part to device memory
	double *d_part;
	err = hipMalloc(&d_part, BLOCK_SIZE*sizeof(double));
	printf("CUDA malloc d_part; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part, DBL_MIN, BLOCK_SIZE*sizeof(double));
	printf("CUDA memset d_part to DBL_MIN: %s\n", hipGetErrorString(err));

	// load d_max to device memory
	double *d_max;
	err = hipMalloc(&d_max, sizeof(double));
	printf("CUDA malloc d_max; %s\n", hipGetErrorString(err));
	err = hipMemset(d_max, DBL_MIN, sizeof(double));
	printf("CUDA memset d_max to DBL_MIN: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((elements + dimBlock.x - 1)/dimBlock.x);
	//int blockDim_2 = NearestPowerOf2(d_A.width*d_A.height);
	//printf("nearest power of 2 (blockDim_2): %d\n",blockDim_2);
	// first pass
	arraySumKernel<<<dimGrid, dimBlock>>>(d_A, elements, d_part);
	err = hipDeviceSynchronize();
	printf("Run kernel 1st pass: %s\n", hipGetErrorString(err));
	// second pass
	dimGrid = dim3(1);
	arraySumKernel<<<dimGrid, dimBlock>>>(d_part, BLOCK_SIZE, d_max);
	err = hipDeviceSynchronize();
	printf("Run kernel 2nd pass: %s\n", hipGetErrorString(err));

	// read max from device memory
	double max;
	err = hipMemcpy(&max, d_max, sizeof(double), hipMemcpyDeviceToHost);
	printf("Copy max off of device: %s\n",hipGetErrorString(err));
	
	// stop the timer
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("Time elapsed: %f ms\n", time);

	// free device memory
	hipFree(d_A);
	hipFree(d_max);
	return max;
}

// matrix populate kernel called by populate()
__global__
void populateKernel(double* d_A, int size) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if(row >= size) return;
	d_A[row] = threadIdx.y; 
}

void populate(double* A, int elements) {
	srand(time(0));
	hipEvent_t start, stop;
	float time;
	// create events and start the timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	// load A to device memory
	double* d_A;
	size_t size = elements * sizeof(double);
	hipError_t err = hipMalloc(&d_A, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	  //invoke kernel
	dim3 dimBlock(BLOCK_SIZE_DIM2, BLOCK_SIZE_DIM2);
	dim3 dimGrid( (elements + dimBlock.x - 1)/dimBlock.x, 1 );
	populateKernel<<<dimGrid, dimBlock>>>(d_A, size);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read A from device memory
	err = hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(err));

	// stop the timer
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("Time elapsed: %f ms\n", time);

	// free device memory
	hipFree(d_A);
}
/*void printMatrix(Matrix A) {
	printf("\n");
	for (int i=0; i<A.height; i++) {
		for (int j=0; j<A.width; j++) {
			printf("%.4f ", A.elements[i*A.width+j]); 
		}
		printf("\n");
	}
	printf("\n");
}*/

//usage : maxOfArray height width
int main(int argc, char* argv[]) {
	double* A;
	int a1;
	// Read some values from the commandline
	a1 = atoi(argv[1]); /* elements in A */
	if (a1 > 1048576) {
		printf("Matrices bigger than 1048576 elements are not supported yet\n");
		return 0;
	}
	A = (double*)malloc(a1 * sizeof(double));
	// give A values
	populate(A, a1);
	//printMatrix(A);
	for(int i=0; i<a1; i++){
		printf("%f \t", A[i]);
	}
	printf("\n");
	// call zeros
	double min = arraySum(A, a1);
	printf("\nThe max element is: %.4f\n", min);
}
