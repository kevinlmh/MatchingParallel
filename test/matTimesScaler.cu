#include "hip/hip_runtime.h"
#include <stdio.h>
#define BLOCK_SIZE 32

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
  int width;
  int height;
	double* elements;
} Matrix;

//function to print a matrix
void printMatrix(Matrix A) {
	printf("\n");
	for (int i=0; i<A.height; i++) {
		for (int j=0; j<A.width; j++) {
			printf("%.4f ", A.elements[i*A.width+j]); 
		}
		printf("\n");
	}
	printf("\n");
}

// matrix matTimesScaler kernel called by matTimesScaler()
__global__
void matTimesScalerKernel(Matrix d_In, double scaler, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	int idx = row * d_In.width +  col;
	d_Out.elements[idx] = d_In.elements[idx] * scaler;
}


void matTimesScaler(Matrix In, double scaler, Matrix Out) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	matTimesScalerKernel<<<dimGrid, dimBlock>>>(d_In, scaler, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// Usage: matTimesScaler
int main(int argc, char* argv[]){
	
	Matrix A, Out;
	Out.width = A.width = 3; 
	Out.height = A.height = 3;
	A.elements = (double*)malloc(A.height*A.width*sizeof(double));
	Out.elements = (double*)malloc(Out.height*Out.width*sizeof(double));
	double AE[3][3] = {{1, 3, 7},{2, 4, 8},{3, 6, 9}};
	memcpy(A.elements, AE, A.height*A.width*sizeof(double));
	
	printf("A:\n");
	printMatrix(A);

	matTimesScaler(A, 2, Out);

	printf("Out:\n");
	printMatrix(Out);
	
	free(A.elements);
	free(Out.elements);
}

