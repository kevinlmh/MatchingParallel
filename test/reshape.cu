#include "hip/hip_runtime.h"
#include <stdio.h>
#define BLOCK_SIZE 32

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
  int width;
  int height;
	double* elements;
} Matrix;

//function to print a matrix
void printMatrix(Matrix A) {
	printf("\n");
	for (int i=0; i<A.height; i++) {
		for (int j=0; j<A.width; j++) {
			printf("%.4f ", A.elements[i*A.width+j]); 
		}
		printf("\n");
	}
	printf("\n");
}

// matrix reshape kernel called by reshape()
__global__
void reshapeKernel(Matrix d_In, Matrix d_Out) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(y >= d_In.height || x >= d_In.width) return;
	int c = x * d_In.height + y;
	d_Out.elements[(c%d_Out.height)*d_Out.width+(c/d_Out.height)] = d_In.elements[(c%d_In.height)*d_In.width+(c/d_In.height)];

}

void reshape(Matrix In, Matrix Out) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
	d_Out.width = Out.width; d_Out.height = Out.height;
	size = Out.width * Out.height * sizeof(double);
	hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	reshapeKernel<<<dimGrid, dimBlock>>>(d_In, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// Usage: reshape
int main(int argc, char* argv[]) {
 
	Matrix A, Out;
	A.height = 3; A.width = 4;
	Out.height = 2; Out.width = 6;
	A.elements = (double*)malloc(A.height*A.width*sizeof(double));
	Out.elements = (double*)malloc(Out.height*Out.width*sizeof(double));
	double AE[3][4] = {{1, 4, 7, 10},{2, 5, 8, 11},{3, 6, 9, 12}};
	memcpy(A.elements, AE, A.height*A.width*sizeof(double));
	
	printf("A:\n");
	printMatrix(A);

	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = d_A.width * d_A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// allocate Out on device memory
	Matrix d_Out;
	d_Out.width = Out.width;
	d_Out.height = Out.height;
	size = d_Out.width * d_Out.height * sizeof(double);
	err = hipMalloc(&d_Out.elements, size);
	printf("CUDA malloc d_Out: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (d_A.width + dimBlock.x - 1)/dimBlock.x, (d_A.height + dimBlock.y - 1)/dimBlock.y );
	reshapeKernel<<<dimGrid, dimBlock>>>(d_A, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy d_Out off of device: %s\n",hipGetErrorString(err));

	printf("Out:\n");
	printMatrix(Out);
	
	// free device memory
	hipFree(d_A.elements);
	hipFree(d_Out.elements);
}

