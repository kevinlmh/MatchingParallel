#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
  int width;
  int height;
	double* elements;
} Matrix;

// matrix zeros kernel called by zeros()
__global__
void reshapeKernel(Matrix d_In, Matrix d_Out) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(y > d_In.height || x > d_In.width) return;
	int c = x * d_In.height + y;
	d_Out.elements[(c%d_Out.height)*d_Out.width+(c/d_Out.height)] = d_In.elements[(c%d_In.height)*d_In.width+(c/d_In.height)];
}

void reshape(Matrix In, Matrix Out) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	reshapeKernel<<<dimGrid, dimBlock>>>(d_In, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

void printMatrix(Matrix A) {
	printf("\n");
	for (int i=0; i<A.height; i++) {
		for (int j=0; j<A.width; j++) {
			printf("%.4f ", A.elements[i*A.width+j]); 
		}
		printf("\n");
	}
	printf("\n");
}

//usage : zeros height width new_height new_width
int main(int argc, char* argv[]) {
	Matrix A, B;
	int a1, a2, b1, b2;
	// Read some values from the commandline
	a1 = atoi(argv[1]); /* Height of A */
	a2 = atoi(argv[2]); /* Width of A */
	b1 = atoi(argv[3]); // Height of B
	b2 = atoi(argv[4]); // Width of B
	if (a1*a2 != b1*b2) {
		printf("Input and output matrices must have the same number of elements");
		return 0;
	}
	A.height = a1;
	A.width = a2;
	B.height = b1;
	B.width = b2;
	A.elements = (double*)malloc(A.width * A.height * sizeof(double));
	B.elements = (double*)malloc(B.width * B.height * sizeof(double));
	// give A random values
	for(int i = 0; i < A.height; i++)
		for(int j = 0; j < A.width; j++)
			A.elements[i*A.width + j] = ((double)rand()/(double)(RAND_MAX)) * 10;
	
	printMatrix(A);
	// call reshape()
	reshape(A, B);
	printMatrix(B);
}
