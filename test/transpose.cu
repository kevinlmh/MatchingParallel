#include "hip/hip_runtime.h"
#include <stdio.h>
#define BLOCK_SIZE 32

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
  int width;
  int height;
	double* elements;
} Matrix;

//function to print a matrix
void printMatrix(Matrix A) {
	printf("\n");
	for (int i=0; i<A.height; i++) {
		for (int j=0; j<A.width; j++) {
			printf("%.4f ", A.elements[i*A.width+j]); 
		}
		printf("\n");
	}
	printf("\n");
}

__global__
void transposeKernel(Matrix d_A, Matrix d_B) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_B.elements[col*d_B.width+row] = d_A.elements[row*d_A.width+col];
}

void transpose(Matrix In, Matrix Out) {
	printf("transpose()\n");
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);

	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);
	printf("Copy In to device: %s\n", hipGetErrorString(err));

	// allocate Out on device memory
	Matrix d_Out;
	d_Out.width = Out.width;
	d_Out.height = Out.height;
	size = d_Out.width * d_Out.height * sizeof(double);
	err = hipMalloc(&d_Out.elements, size);
	printf("CUDA malloc d_Out: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	transposeKernel<<<dimGrid, dimBlock>>>(d_In, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy d_Out off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// Usage: reshape
int main(int argc, char* argv[]) {
 
	Matrix A, At;
	A.height = 3; A.width = 4;
	At.height = A.width; At.width = A.height;
	A.elements = (double*)malloc(A.height*A.width*sizeof(double));
	At.elements = (double*)malloc(At.height*At.width*sizeof(double));
	double AE[3][4] = {	{1, 4, 7, 10},
											{2, 5, 8, 11},
											{3, 6, 9, 12}
										};
	memcpy(A.elements, AE, A.height*A.width*sizeof(double));
	
	printf("A:\n");
	printMatrix(A);
	
	transpose(A, At);
	
	printf("At:\n");
	printMatrix(At);
	
	// free device memory
	free(A.elements);
	free(At.elements);
}

