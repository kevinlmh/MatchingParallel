#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
	int width;
	int height;
	double* elements;
} Matrix;

//matrix transpose kernel called by transpose()
__global__
void transposeKernel(Matrix d_A, Matrix d_B){
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_B.elements[col*d_B.width + row] = d_A.elements[row*d_A.width + col];
}

void transpose(Matrix A, Matrix B){

// load A and B to device memory
	Matrix d_A;
	Matrix d_B;
	d_A.width = A.width;
	d_B.width = B.width;
	d_A.height = A.height;
	d_B.height = B.height;
	size_t size = A.width * A.height * sizeof(double);

	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	hipError_t errB = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(errB));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(errB));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	transposeKernel<<<dimGrid, dimBlock>>>(d_A, d_B);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n",hipGetErrorString(err));
	errB = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy off of device: %s\n", hipGetErrorString(errB));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
}

void printMatrix(Matrix A) {
	printf("\n");
	for (int i=0; i<A.height; i++) {
		for (int j=0; j<A.width; j++) {
			printf("%.4f ", A.elements[i*A.width+j]); 
		}
		printf("\n");
	}
	printf("\n");
}

//usage : tranpose height width
int main(int argc, char* argv[]) {

	srand(time(0));

	Matrix A;
	Matrix B;
	int a1, a2;
	// Read some values from the commandline
	a1 = atoi(argv[1]); /* Height of A */
	a2 = atoi(argv[2]); /* Width of A */
	A.height = a1;
	A.width = a2;
	A.elements = (double*)malloc(A.width * A.height * sizeof(double));
	B.height = a2;
	B.width = a1;
	B.elements = (double*)malloc(B.width * B.height * sizeof(double));
	// give A random values
	for(int i = 0; i < A.height; i++)
		for(int j = 0; j < A.width; j++)
			A.elements[i*A.width + j] = ((double)rand()/(double)(RAND_MAX)) * 10;

	// call tranpose
	transpose(A, B);
	printMatrix(A);
	printMatrix(B);
}
