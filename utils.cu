#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <limits.h>

#pragma once
#define BLOCK_SIZE 32
#define BLOCK_SIZE_DIM1 1024

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
  int width;
  int height;
	float* elements;
} Matrix;

//function to print a matrix
void printMatrix(Matrix A) {
	printf("\n");
	for (int i=0; i<A.height; i++) {
		for (int j=0; j<A.width; j++) {
			printf("%.4f ", A.elements[i*A.width+j]); 
		}
		printf("\n");
	}
	printf("\n");
}

// matrix zeros kernel called by zeros()
__global__
void zerosKernel(Matrix d_A) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_A.elements[row*d_A.width+col] = 0;
}

void zeros(Matrix A) {
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	zerosKernel<<<dimGrid, dimBlock>>>(d_A);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
}

//matrix transpose kernel
__global__
void transposeKernel(Matrix d_A, Matrix d_B){
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_B.elements[col*d_B.width + row] = d_A.elements[row*d_A.width + col];
}

void transpose(Matrix In, Matrix Out){
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(float);

	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);
	printf("Copy In to device: %s\n", hipGetErrorString(err));

	// allocate Out on device memory
	Matrix d_Out;
	d_Out.width = Out.width;
	d_Out.height = Out.width;
	size = d_Out.width * d_Out.height * sizeof(float);
	err = hipMalloc(&d_Out.elements, size);
	printf("CUDA malloc d_Out: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	transposeKernel<<<dimGrid, dimBlock>>>(d_In, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy d_Out off of device: %s\n",hipGetErrorString(err));

// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// matrix ones kernel called by ones()
__global__
void onesKernel(Matrix d_A) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	d_A.elements[row*d_A.width+col] = 1;
}

void ones(Matrix A) {
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	onesKernel<<<dimGrid, dimBlock>>>(d_A);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
}

// check if a square matrix is symmetric
__global__
void isSymmetricKernel(Matrix d_A, int *d_result) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	if(d_A.elements[row*d_A.width+col] != d_A.elements[row + col*d_A.width])
		*(d_result) = 0;
}

int isSymmetric(Matrix A) {
	printf("isSymmetric()\n");
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load result to device memory
	int result = 1;
	int *d_result;
	err = hipMalloc(&d_result, sizeof(int));
	printf("CUDA malloc d_result: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);	
	printf("Copy result to device: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	isSymmetricKernel<<<dimGrid, dimBlock>>>(d_A, d_result);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	//read result from fdevice memory
	err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
	printf("Copy result off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_result);

	return result;
}

// check if a matrix is symmetric
__global__
void isSymmetricEpsKernel(Matrix d_A, int *d_result, float eps){

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	if(d_A.elements[row*d_A.width+col] + eps < d_A.elements[row + col*d_A.width] && 
		d_A.elements[row*d_A.width+col] - eps > d_A.elements[row + col*d_A.width])
	
		*(d_result) = 0;
}

int isSymmetricEps(Matrix A, float eps) {
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load result to device memory
	int result = 1;
	int *d_result;
	err = hipMalloc(&d_result, sizeof(int));
	printf("CUDA malloc d_result: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);	
	printf("Copy result to device: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	isSymmetricEpsKernel<<<dimGrid, dimBlock>>>(d_A, d_result, eps);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read result from device memory
	err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
	printf("Copy result off of device: %s\n",hipGetErrorString(err));


	// free device memory
	hipFree(d_A.elements);
	hipFree(d_result);
	
	return result;

}

//create an m-by-n tiling of a given matrix
__global__
void repmatKernel(Matrix d_A, int m, int n, Matrix d_B) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	for(int i=0; i < m; i++) {
		for(int j=0; j < n; j++) {
			d_B.elements[(row + i*d_A.height)*d_B.width + (col + j*d_A.width)] = d_A.elements[row*d_A.width + col];
		}
	}
}

void repmat(Matrix In, int m, int n, Matrix Out){
	if (Out.height != In.height * m || Out.width != In.width * n) {
		printf("Output matrix has incorrect dimensions!\n");
		return;
	}
	// load In  to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = d_In.width * d_In.height * sizeof(float);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);
	printf("Copy In to device: %s\n", hipGetErrorString(err));

	// allocate Out on device memory
	Matrix d_Out;
	d_Out.width = In.width * n;
	d_Out.height = In.height * m;
	size = d_Out.width * d_Out.height * sizeof(float);
	err = hipMalloc(&d_Out.elements, size);
	printf("CUDA malloc d_Out: %s\n", hipGetErrorString(err));
	hipMemcpy(d_Out.elements, Out.elements, size, hipMemcpyHostToDevice);
	printf("Copy d_Out to device: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	repmatKernel<<<dimGrid, dimBlock>>>(d_In, m, n, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy d_Out off of device: %s\n",hipGetErrorString(err));

// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// matSub kernel
__global__
void matSubKernel(Matrix d_A, Matrix d_B, Matrix d_C) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_C.elements[row*d_A.width + col] = d_A.elements[row*d_A.width + col] - d_B.elements[row*d_A.width + col];
}

void matSub(Matrix A, Matrix B, Matrix C){

	// load A, B to device memory
	Matrix d_A;
	Matrix d_B;
	d_A.width = A.width;
	d_B.width = B.width;
	d_A.height = A.height;
	d_B.height = B.height;
	size_t size = A.width * A.height * sizeof(float);

	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(err));
	
	// allocate C to device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	err = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matSubKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read C from device memory
	err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n", hipGetErrorString(err));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}


// matAdd kernel
__global__
void matAddKernel(Matrix d_A, Matrix d_B, Matrix d_C) {
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_C.elements[row*d_C.width + col] = d_A.elements[row*d_A.width + col] + d_B.elements[row*d_B.width + col];
}


void matAdd(Matrix A, Matrix B, Matrix C){

	// load A, B to device memory
	Matrix d_A;
	Matrix d_B;
	d_A.width = A.width;
	d_B.width = B.width;
	d_A.height = A.height;
	d_B.height = B.height;
	size_t size = A.width * A.height * sizeof(float);

	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(err));
	
	// allocate C on device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	err = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matAddKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read C from device memory
	err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n", hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

// matrix matTimesScaler kernel called by matTimesScaler()
__global__
void matTimesScalerKernel(Matrix d_In, float scaler, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	int idx = row * d_In.width +  col;
	d_Out.elements[idx] = d_In.elements[idx] * scaler;
}


void matTimesScaler(Matrix In, float scaler, Matrix Out) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(float);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(float);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	matTimesScalerKernel<<<dimGrid, dimBlock>>>(d_In, scaler, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// matrix matPlusScaler kernel called by matPlusScaler()
__global__
void matPlusScalerKernel(Matrix d_In, float scaler, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	int idx = row * d_In.width +  col;
	d_Out.elements[idx] = d_In.elements[idx] + scaler;
}

void matPlusScaler(Matrix In, float scaler, Matrix Out) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(float);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(float);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	matPlusScalerKernel<<<dimGrid, dimBlock>>>(d_In, scaler, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);

}

// matrix matDiv kernel called by matDiv()
__global__
void matDivKernel(Matrix d_A, Matrix d_B, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row*d_A.width+col;
	if(row > d_A.height || col > d_A.width) return;
	d_Out.elements[idx] = d_A.elements[idx] / d_B.elements[idx];
}

void matDiv(Matrix A, Matrix B, Matrix Out) {
	if (A.width != B.width || A.height != B.height) {
		printf("Input matrices must have the same dimension!\n");
		return;
	}
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix A to device: %s\n", hipGetErrorString(err));
	
	// load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix B to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(float);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matDivKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_Out.elements);

}


// matrix getCol kernel
__global__
void getColKernel(Matrix d_In, Matrix d_Out, int num) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	if(col == num) 
		d_Out.elements[row] = d_In.elements[row*d_In.width+col];
}

void getCol(Matrix In, Matrix Out, int num) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(float);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(float);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	getColKernel<<<dimGrid, dimBlock>>>(d_In, d_Out, num);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output row off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// matrix getRow kernel
__global__
void getRowKernel(Matrix d_In, Matrix d_Out, int num) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	if(row == num) 
		d_Out.elements[col] = d_In.elements[row*d_In.width+col];
}

void getRow(Matrix In, Matrix Out, int num) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(float);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(float);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	getRowKernel<<<dimGrid, dimBlock>>>(d_In, d_Out, num);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output row off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}


// matrix indexOfElement kernel
__global__
void indexOfElementKernel(Matrix d_A, float element, int *index) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	int idx = row*d_A.width+col;
	if (d_A.elements[idx] == element)
		*(index) = idx;
}

int indexOfElement(Matrix d_A, float element) {
	int index = -1;	

	// allocate d_index on device memory
	int *d_index;
	hipError_t err = hipMalloc(&d_index, sizeof(int));
	printf("CUDA malloc index; %s\n", hipGetErrorString(err));
	hipMemcpy(d_index, &index, sizeof(int), hipMemcpyHostToDevice);
	printf("Copy index to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (d_A.width + dimBlock.x - 1)/dimBlock.x, (d_A.height + dimBlock.y - 1)/dimBlock.y );
	indexOfElementKernel<<<dimGrid, dimBlock>>>(d_A, element, d_index);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read index from device memory
	err = hipMemcpy(&index, d_index, sizeof(int), hipMemcpyDeviceToHost);
	printf("Copy index off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_index);
	return index;
}


// matrix reshape kernel called by reshape()
__global__
void reshapeKernel(Matrix d_In, Matrix d_Out) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(y > d_In.height || x > d_In.width) return;
	int c = x * d_In.height + y;
	d_Out.elements[(c%d_Out.height)*d_Out.width+(c/d_Out.height)] = d_In.elements[(c%d_In.height)*d_In.width+(c/d_In.height)];
}

void reshape(Matrix In, Matrix Out) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(float);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
	d_Out.width = Out.width; d_Out.height = Out.height;
	size = Out.width * Out.height * sizeof(float);
	hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	reshapeKernel<<<dimGrid, dimBlock>>>(d_In, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

__global__
void maxReduceKernel(float *elements, int size, float *d_part) {
	int  thread2;
	float temp;
	__shared__ float sdata[BLOCK_SIZE_DIM1];
	
	// Load max from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		sdata[threadIdx.x] = elements[idx];
	else
		sdata[threadIdx.x] = DBL_MIN;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE_DIM1;
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;
			// Get the shared value stored by another thread 
			temp = sdata[thread2];
			if (temp > sdata[threadIdx.x]) 
				 sdata[threadIdx.x] = temp;
		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the max to d_max
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

float maxOfMatrix(Matrix d_A) {

	// allocate d_part1 on device memory
	float *d_part1;
	hipError_t err = hipMalloc(&d_part1, BLOCK_SIZE_DIM1*BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA malloc d_part1; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part1, DBL_MIN,  BLOCK_SIZE_DIM1*BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA memset d_part1 to DBL_MIN: %s\n", hipGetErrorString(err));	
	
	// allocate d_part2 on device memory
	float *d_part2;
	err = hipMalloc(&d_part2, BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA malloc d_part2; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part1, DBL_MIN, BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA memset d_part2 to DBL_MIN: %s\n", hipGetErrorString(err));	
	
	// allocate d_max on device memory
	float *d_max;
	err = hipMalloc(&d_max, sizeof(float));
	printf("CUDA malloc d_max; %s\n", hipGetErrorString(err));
	err = hipMemset(d_max, DBL_MIN, sizeof(float));
	printf("CUDA memset d_max to DBL_MIN: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE_DIM1);
	dim3 dimGrid((d_A.width*d_A.height + dimBlock.x - 1)/dimBlock.x);
	
	// first pass
	maxReduceKernel<<<dimGrid, dimBlock>>>(d_A.elements, d_A.width*d_A.height, d_part1);
	err = hipDeviceSynchronize();
	printf("Run kernel 1st pass: %s\n", hipGetErrorString(err));
	
	// second pass
	dimGrid = dim3(BLOCK_SIZE_DIM1);
	maxReduceKernel<<<dimGrid, dimBlock>>>(d_part1, BLOCK_SIZE_DIM1*BLOCK_SIZE_DIM1, d_part2);
	err = hipDeviceSynchronize();
	printf("Run kernel 2nd pass: %s\n", hipGetErrorString(err));
	
	// third pass
	dimGrid = dim3(1);
	maxReduceKernel<<<dimGrid, dimBlock>>>(d_part2, BLOCK_SIZE_DIM1, d_max);
	err = hipDeviceSynchronize();
	printf("Run kernel 3rd pass: %s\n", hipGetErrorString(err));

	// read max from device memory
	float max;
	err = hipMemcpy(&max, d_max, sizeof(float), hipMemcpyDeviceToHost);
	printf("Copy max off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_part1);
	hipFree(d_part2);
	hipFree(d_max);
	
	return max;
}

__global__
void minReduceKernel(float *elements, int size, float *d_part) {
	int  thread2;
	float temp;
	__shared__ float sdata[BLOCK_SIZE_DIM1];
	
	// Load data from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		sdata[threadIdx.x] = elements[idx];
	else
		sdata[threadIdx.x] = DBL_MAX;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE_DIM1;
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;
			// Get the shared value stored by another thread 
			temp = sdata[thread2];
			if (temp < sdata[threadIdx.x]) 
				 sdata[threadIdx.x] = temp;

		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the min to d_min
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

float minOfMatrix(Matrix d_A) {

	// allocate d_part1 on device memory
	float *d_part1;
	hipError_t err = hipMalloc(&d_part1, BLOCK_SIZE_DIM1*BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA malloc d_part1; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part1, DBL_MAX,  BLOCK_SIZE_DIM1*BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA memset d_part1 to DBL_MAX: %s\n", hipGetErrorString(err));	
	
	// allocate d_part2 on device memory
	float *d_part2;
	err = hipMalloc(&d_part2, BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA malloc d_part2; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part1, DBL_MAX, BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA memset d_part2 to DBL_MAX: %s\n", hipGetErrorString(err));	
	
	// allocate d_min on device memory
	float *d_min;
	err = hipMalloc(&d_min, sizeof(float));
	printf("CUDA malloc d_min; %s\n", hipGetErrorString(err));
	err = hipMemset(d_min, DBL_MAX, sizeof(float));
	printf("CUDA memset d_min to DBL_MAX: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE_DIM1);
	dim3 dimGrid((d_A.width*d_A.height + dimBlock.x - 1)/dimBlock.x);
	
	// first pass
	minReduceKernel<<<dimGrid, dimBlock>>>(d_A.elements, d_A.width*d_A.height, d_part1);
	err = hipDeviceSynchronize();
	printf("Run kernel 1st pass: %s\n", hipGetErrorString(err));
	
	// second pass
	dimGrid = dim3(BLOCK_SIZE_DIM1);
	minReduceKernel<<<dimGrid, dimBlock>>>(d_part1, BLOCK_SIZE_DIM1*BLOCK_SIZE_DIM1, d_part2);
	err = hipDeviceSynchronize();
	printf("Run kernel 2nd pass: %s\n", hipGetErrorString(err));
	
	// third pass
	dimGrid = dim3(1);
	minReduceKernel<<<dimGrid, dimBlock>>>(d_part2, BLOCK_SIZE_DIM1, d_min);
	err = hipDeviceSynchronize();
	printf("Run kernel 3rd pass: %s\n", hipGetErrorString(err));

	// read max from device memory
	float min;
	err = hipMemcpy(&min, d_min, sizeof(float), hipMemcpyDeviceToHost);
	printf("Copy min off of device: %s\n",hipGetErrorString(err));
	
	// free device memory
	hipFree(d_part1);
	hipFree(d_part2);
	hipFree(d_min);
	
	return min;
}


__global__
void sumReduceKernel(float *elements, int size, float *d_part) {
	int  thread2;
	__shared__ float sdata[BLOCK_SIZE_DIM1];
	
	// Load elements from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		sdata[threadIdx.x] = elements[idx];
	else
		sdata[threadIdx.x] = 0;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE_DIM1;
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;
			// Get the shared value stored by another thread and sum it to sdata
			sdata[threadIdx.x] += sdata[thread2];

		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the max to d_max
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

float matSum(Matrix d_A) {

	// allocate d_part1 on device memory
	float *d_part1;
	hipError_t err = hipMalloc(&d_part1, BLOCK_SIZE_DIM1*BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA malloc d_part1; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part1, 0,  BLOCK_SIZE_DIM1*BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA memset d_part1 to 0: %s\n", hipGetErrorString(err));	
	
	// allocate d_part2 on device memory
	float *d_part2;
	err = hipMalloc(&d_part2, BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA malloc d_part2; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part1, 0, BLOCK_SIZE_DIM1*sizeof(float));
	printf("CUDA memset d_part2 to 0: %s\n", hipGetErrorString(err));	
	
	// allocate d_sum on device memory
	float *d_sum;
	err = hipMalloc(&d_sum, sizeof(float));
	printf("CUDA malloc d_sum; %s\n", hipGetErrorString(err));
	err = hipMemset(d_sum, 0, sizeof(float));
	printf("CUDA memset d_sum to 0: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE_DIM1);
	dim3 dimGrid((d_A.width*d_A.height + dimBlock.x - 1)/dimBlock.x);
	
	// first pass
	sumReduceKernel<<<dimGrid, dimBlock>>>(d_A.elements, d_A.width*d_A.height, d_part1);
	err = hipDeviceSynchronize();
	printf("Run kernel 1st pass: %s\n", hipGetErrorString(err));
	
	// second pass
	dimGrid = dim3(BLOCK_SIZE_DIM1);
	sumReduceKernel<<<dimGrid, dimBlock>>>(d_part1, BLOCK_SIZE_DIM1*BLOCK_SIZE_DIM1, d_part2);
	err = hipDeviceSynchronize();
	printf("Run kernel 2nd pass: %s\n", hipGetErrorString(err));
	
	// third pass
	dimGrid = dim3(1);
	sumReduceKernel<<<dimGrid, dimBlock>>>(d_part2, BLOCK_SIZE_DIM1, d_sum);
	err = hipDeviceSynchronize();
	printf("Run kernel 3rd pass: %s\n", hipGetErrorString(err));

	// read sum from device memory
	float sum;
	err = hipMemcpy(&sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
	printf("Copy sum off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_part1);
	hipFree(d_part2);
	hipFree(d_sum);
	
	return sum;
}


__global__
void maxOfMatrixRow(Matrix d_A, Matrix d_col) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	float max = d_A.elements[row*d_A.width];
	for (int col=0; col<d_A.width; col++) {
		max = (d_A.elements[row*d_A.width+col] > max)? d_A.elements[row*d_A.width+col] : max;
	}
	d_col.elements[row] = max;
}


__global__
void maxOfMatrixCol(Matrix d_A, Matrix d_row) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float max = d_A.elements[col];
	for (int row=0; row<d_A.height; row++) {
		max = (d_A.elements[row*d_A.width+col] > max)? d_A.elements[row*d_A.width+col] : max;
	}
	d_row.elements[col] = max;
}

__global__
void sumOfMatrixRowKernel(Matrix d_In, Matrix d_sumCol) {
	int idx =  blockIdx.x * blockDim.x + threadIdx.x;	
	dim3 dimBlock(BLOCK_SIZE_DIM1);
	dim3 dimGrid( (d_In.width + dimBlock.x - 1)/dimBlock.x );
	
	// two pass sum reduction
	// allocate d_part
	float *d_part = (float*)malloc(dimGrid.x * sizeof(float));
	// allocate d_sum 
	float *d_sum = (float*)malloc(sizeof(float));
	memset(d_sum, 0, sizeof(float));

	// first pass
	sumReduceKernel<<<dimGrid, dimBlock>>>(d_In.elements, d_In.width, d_part);
	
	// second pass
	dimGrid = dim3(1);
	sumReduceKernel<<<dimGrid, dimBlock>>>(d_part, dimGrid.x, d_sum);

	// write d_sum to d_sumCol
	d_sumCol.elements[idx] = *d_sum;

	// free device memory
	free(d_part);
}

void sumOfMatrixRow(Matrix In, Matrix sumCol) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(float);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate sumCol in device memory
	Matrix d_sumCol;
	d_sumCol.width = sumCol.width; d_sumCol.height = sumCol.height;
	size = sumCol.width * sumCol.height * sizeof(float);
	err = hipMalloc(&d_sumCol.elements, size);
	printf("CUDA malloc sumCol: %s\n", hipGetErrorString(err));
	
	// lauch one thread for each row to do the sum
	dim3 dimBlock(BLOCK_SIZE_DIM1);
	dim3 dimGrid( (In.height + dimBlock.x - 1)/dimBlock.x );
	sumOfMatrixRowKernel<<<dimGrid, dimBlock>>>(d_In, d_sumCol);
	err = hipDeviceSynchronize();
	printf("Run sum of matrix kernel: %s\n", hipGetErrorString(err));
	
	// read sumCol from device memory
	err = hipMemcpy(sumCol.elements, d_sumCol.elements, size, hipMemcpyDeviceToHost);
	printf("Copy sumCol off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_sumCol.elements);
}

__global__
void colSumReduceKernel(Matrix d_In, int colNum, float *d_part) {
	int  thread2;
	__shared__ float sdata[BLOCK_SIZE_DIM1];
	
	// Load elements from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < d_In.height)
		sdata[threadIdx.x] = d_In.elements[idx*d_In.width + colNum];
	else
		sdata[threadIdx.x] = 0;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE_DIM1;
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;
			// Get the shared value stored by another thread and sum it to sdata
			sdata[threadIdx.x] += sdata[thread2];

		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the max to d_max
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

__global__
void sumOfMatrixColKernelOne(Matrix d_In, Matrix d_sumRow) {
	int idx =  blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= d_In.width) return;
	
	dim3 dimBlock(BLOCK_SIZE_DIM1);
	dim3 dimGrid( (d_In.height + dimBlock.x - 1)/dimBlock.x );
	
	// two pass sum reduction
	// allocate d_part
	float *d_part = (float*)malloc(dimGrid.x * sizeof(float));
	memset(d_part, 0, dimGrid.x * sizeof(float));
	// allocate d_sum 
	float *d_sum = (float*)malloc(sizeof(float));
	memset(d_sum, 0, sizeof(float));

	// first pass
	colSumReduceKernel<<<dimGrid, dimBlock>>>(d_In, idx, d_part);
	
	// second pass
	dimGrid = dim3(1);
	sumReduceKernel<<<dimGrid, dimBlock>>>(d_part, dimGrid.x, d_sum);

	// write d_sum to d_sumCol
	d_sumRow.elements[idx] = *d_sum;

	// free device memory
	free(d_part);
}

void sumOfMatrixCol(Matrix In, Matrix sumRow) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(float);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate sumRow in device memory
	Matrix d_sumRow;
	d_sumRow.width = sumRow.width; d_sumRow.height = sumRow.height;
	size = sumRow.width * sumRow.height * sizeof(float);
	err = hipMalloc(&d_sumRow.elements, size);
	printf("CUDA malloc sumrow: %s\n", hipGetErrorString(err));
	
	// lauch one thread for each col to do the sum
	dim3 dimBlock(BLOCK_SIZE_DIM1);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x );
	sumOfMatrixRowKernel<<<dimGrid, dimBlock>>>(d_In, d_sumRow);
	err = hipDeviceSynchronize();
	printf("Run sum of matrix row kernel: %s\n", hipGetErrorString(err));
	
	// read sumRow from device memory
	err = hipMemcpy(sumRow.elements, d_sumRow.elements, size, hipMemcpyDeviceToHost);
	printf("Copy sumRow off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_sumRow.elements);
}
