#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>

#define BLOCK_SIZE 16

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
  int width;
  int height;
	double* elements;
} Matrix;

//function to print a matrix
void printMatrix(Matrix A) {
	printf("\n");
	for (int i=0; i<A.height; i++) {
		for (int j=0; j<A.width; j++) {
			printf("%.4f ", A.elements[i*A.width+j]); 
		}
		printf("\n");
	}
	printf("\n");
}

// matrix zeros kernel called by zeros()
__global__
void zerosKernel(Matrix d_A) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	d_A.elements[row*d_A.width+col] = 0;
}

void zeros(Matrix A) {
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	zerosKernel<<<dimGrid, dimBlock>>>(d_A);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
}

//matrix transpose kernel called by transpose()
__global__
void transposeKernel(Matrix d_A, Matrix d_B){
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_B.elements[col*d_B.width + row] = d_A.elements[row*d_A.width + col];
}

void transpose(Matrix A, Matrix B){

// load A and B to device memory
	Matrix d_A;
	Matrix d_B;
	d_A.width = A.width;
	d_B.width = B.width;
	d_A.height = A.height;
	d_B.height = B.height;
	size_t size = A.width * A.height * sizeof(double);

	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	hipError_t errB = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(errB));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(errB));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	transposeKernel<<<dimGrid, dimBlock>>>(d_A, d_B);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n",hipGetErrorString(err));
	errB = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy off of device: %s\n", hipGetErrorString(errB));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
}

// matrix ones kernel called by ones()
__global__
void onesKernel(Matrix d_A) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	d_A.elements[row*d_A.width+col] = 1;
}

void ones(Matrix A) {
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	zerosKernel<<<dimGrid, dimBlock>>>(d_A);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
}

// check if a matrix is symmetric
__global__
void isSymmetricKernel(Matrix d_A, Matrix d_B){

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	if(d_A.elements[row*d_A.width+col] == d_A.elements[row + col*d_A.width])
		return;
	else
		d_B.elements[row*d_B.width + col] = 1;
}

void isSymmetric(Matrix A, Matrix B) {
// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t errA = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(errA));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(errA));

// load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	hipError_t errB = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(errB));	
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(errB));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	isSymmetricKernel<<<dimGrid, dimBlock>>>(d_A, d_B);
	hipError_t err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	errA = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(errA));

//read B from device memory
	errB = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(errB));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
}

// check if a matrix is symmetric
__global__
void isSymmetricEpsKernel(Matrix d_A, Matrix d_B, double eps){

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	if(d_A.elements[row*d_A.width+col] + eps >= d_A.elements[row + col*d_A.width] || 
		d_A.elements[row*d_A.width+col] - eps <= d_A.elements[row + col*d_A.width])
		return;
	else
		d_B.elements[row*d_B.width + col] = 1;
}

void isSymmetricEps(Matrix A, Matrix B, double eps) {
// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t errA = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(errA));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(errA));

// load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	hipError_t errB = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(errB));	
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(errB));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	isSymmetricEpsKernel<<<dimGrid, dimBlock>>>(d_A, d_B, eps);
	hipError_t err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	errA = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(errA));

//read B from device memory
	errB = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(errB));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
}

//create an m-by-n tiling of a given matrix
__global__
void repmatKernel(Matrix d_A, Matrix d_B){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	int h_reps = d_B.width / d_A.width;
	int v_reps = d_B.height / d_A.height;
	for(int i=0; i < h_reps; i++){
		for(int j=0; j < v_reps; j++){
			d_B.elements[row*d_B.width + col + d_A.width*i + d_B.width*j*d_A.height] = d_A.elements[row*d_A.width + col];
		}
	}
}

void repmat(Matrix A, Matrix B){
// load A and B to device memory
	Matrix d_A;
	Matrix d_B;
	d_A.width = A.width;
	d_B.width = B.width;
	d_A.height = A.height;
	d_B.height = B.height;
	size_t sizeA = A.width * A.height * sizeof(double);
	size_t sizeB = B.width * B.height * sizeof(double);

	hipError_t err = hipMalloc(&d_A.elements, sizeA);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	hipMemcpy(d_A.elements, A.elements, sizeA, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	hipError_t errB = hipMalloc(&d_B.elements, sizeB);
	printf("CUDA malloc B: %s\n", hipGetErrorString(errB));
	hipMemcpy(d_B.elements, B.elements, sizeB, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(errB));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	repmatKernel<<<dimGrid, dimBlock>>>(d_A, d_B);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, sizeA, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(err));
	err = hipMemcpy(B.elements, d_B.elements, sizeB, hipMemcpyDeviceToHost);
	printf("Copy B off of device: %s\n", hipGetErrorString(errB));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
}

__global__
void matSubKernel(Matrix d_A, Matrix d_B, Matrix d_C){
int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	d_C.elements[row*A.width + col] = d_A.elements[row*A.width + col] - d_B.elements[row*A.width + col];
}

void matSub(Matrix A, Matrix B, Matrix C){

// load A, B, and C to device memory
	Matrix d_A;
	Matrix d_B;
	Matrix d_C;
	d_A.width = A.width;
	d_B.width = B.width;
	d_C.width = C.width;
	d_A.height = A.height;
	d_B.height = B.height;
	d_C.height = C.height;
	size_t size = A.width * A.height * sizeof(double);

	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	hipError_t err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(err));

	hipError_t err = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));
	hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);
	printf("Copy C to device: %s\n", hipGetErrorString(err));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matSubKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(err));
	err = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy B off of device: %s\n", hipGetErrorString(err));
	err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n", hipGetErrorString(err));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

__global__
void matAddKernel(Matrix d_A, Matrix d_B, Matrix d_C){

	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_C.elements[row*d_C.width + col] = d_A.elements[row*d_A.width + col] + d_B.elements[row*d_B.width + col];
}

void matAdd(Matrix A, Matrix B, Matrix C){

// load A, B, and C to device memory
	Matrix d_A;
	Matrix d_B;
	Matrix d_C;
	d_A.width = A.width;
	d_B.width = B.width;
	d_C.width = C.width;
	d_A.height = A.height;
	d_B.height = B.height;
	d_C.height = C.height;
	size_t size = A.width * A.height * sizeof(double);

	hipError_t errA = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(errA));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(errA));

	hipError_t errB = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(errB));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(errB));

	hipError_t errC = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(errC));
	hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);
	printf("Copy C to device: %s\n", hipGetErrorString(errC));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matAddKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipError_t err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(errA));
	err = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy B off of device: %s\n", hipGetErrorString(errB));
	err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n", hipGetErrorString(errC));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}


__global__
void matTimesScalerKernel(Matrix d_A, Matrix d_B, double C){
int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	d_B.elements[row*d_A.width + col] = d_A.elements[row*d_A.width + col] * C;
}

void matTimesScaler(Matrix A, Matrix B, double C){

// load A, B, and C to device memory
	Matrix d_A;
	Matrix d_B;
	
	d_A.width = A.width;
	d_B.width = B.width;
	d_A.height = A.height;
	d_B.height = B.height;
	size_t size = A.width * A.height * sizeof(double);

	hipError_t errA = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(errA));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(errA));

	hipError_t errB = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(errB));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(errB));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matTimesScalerKernel<<<dimGrid, dimBlock>>>(d_A, d_B, C);
	hipError_t err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	errA = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(errA));
// read B from device memory
	errB = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy B off of device: %s\n", hipGetErrorString(errB));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
}

__global__
void matPlusScalerKernel(Matrix d_A, Matrix d_B, double C){
int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	d_B.elements[row*d_A.width + col] = d_A.elements[row*d_A.width + col] + C;
}

void matPlusScaler(Matrix A, Matrix B, double C){

// load A, B, and C to device memory
	Matrix d_A;
	Matrix d_B;
	
	d_A.width = A.width;
	d_B.width = B.width;
	d_A.height = A.height;
	d_B.height = B.height;
	size_t size = A.width * A.height * sizeof(double);

	hipError_t errA = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(errA));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(errA));

	hipError_t errB = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(errB));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(errB));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matPlusScalerKernel<<<dimGrid, dimBlock>>>(d_A, d_B, C);
	hipError_t err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	errA = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(errA));
// read B from device memory
	errB = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy B off of device: %s\n", hipGetErrorString(errB));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
}

__global__
void matDivKernel(Matrix d_A, Matrix d_B, Matrix d_C){
int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	d_C.elements[row*d_C.width + col] = d_A.elements[row*d_A.width + col] / d_B.elements[row*d_B.width + col];
}

void matDiv(Matrix A, Matrix B, Matrix C){

// load A, B, and C to device memory
	Matrix d_A;
	Matrix d_B;
	Matrix d_C;
	d_A.width = A.width;
	d_B.width = B.width;
	d_C.width = C.width;
	d_A.height = A.height;
	d_B.height = B.height;
	d_C.height = C.height;
	size_t size = A.width * A.height * sizeof(double);

	hipError_t errA = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(errA));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(errA));

	hipError_t errB = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(errB));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(errB));

	hipError_t errC = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(errC));
	hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);
	printf("Copy C to device: %s\n", hipGetErrorString(errC));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matDivKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipError_t err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	errA = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(errA));
// read B from device memory
	errB = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy B off of device: %s\n", hipGetErrorString(errB));
// read C from device memory
	errC = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n", hipGetErrorString(errC));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

// matrix zeros kernel called by getCol()
__global__
void getColKernel(Matrix d_In, Matrix d_Out, int num) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	if(col == num) 
		d_Out.elements[row] = d_In.elements[row*d_In.width+col];
}

void getCol(Matrix In, Matrix Out, int num) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	getColKernel<<<dimGrid, dimBlock>>>(d_In, d_Out, num);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output row off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// matrix zeros kernel called by getRow()
__global__
void getRowKernel(Matrix d_In, Matrix d_Out, int num) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	if(row == num) 
		d_Out.elements[col] = d_In.elements[row*d_In.width+col];
}

void getRow(Matrix In, Matrix Out, int num) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	getRowKernel<<<dimGrid, dimBlock>>>(d_In, d_Out, num);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output row off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// matrix zeros kernel called by indexOfElement()
__global__
void indexOfElementKernel(Matrix d_A, double element, int *index) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	int idx = row*d_A.width+col;
	if (d_A.elements[idx] == element)
		*(index) = idx;
}

int indexOfElement(Matrix A, double element) {
	int index;	
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load index to device memory
	int *d_index;
	hipMemset(d_index, -1, sizeof(int));
	err = hipMalloc(&d_index, sizeof(int));
	printf("CUDA malloc index; %s\n", hipGetErrorString(err));
	hipMemcpy(d_index, &index, sizeof(int), hipMemcpyHostToDevice);
	printf("Copy index to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	indexOfElementKernel<<<dimGrid, dimBlock>>>(d_A, element, d_index);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read index from device memory
	err = hipMemcpy(&index, d_index, sizeof(int), hipMemcpyDeviceToHost);
	printf("Copy index off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_index);
	return index;
}

__global__
void maxReduceKernel(double *elements, int size, double *d_part) {
	// Reduction max, works for any blockDim.x:
	int  thread2;
	double temp;
	__shared__ double sdata[BLOCK_SIZE];
	
	// Load max from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		sdata[threadIdx.x] = elements[idx];
	else
		sdata[threadIdx.x] = DBL_MIN;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE;	// Total number of threads, rounded up to the next power of two
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;

			// Skipping the fictious threads blockDim.x ... blockDim_2-1
			if (thread2 < blockDim.x) {
				// Get the shared value stored by another thread 
				temp = sdata[thread2];
				if (temp > sdata[threadIdx.x]) 
					 sdata[threadIdx.x] = temp;
			}
		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the max to d_max
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

double maxOfMatrix(Matrix A) {
	hipEvent_t start, stop;
	float time;
	// create events and start the timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load d_part to device memory
	double *d_part;
	err = hipMalloc(&d_part, BLOCK_SIZE*sizeof(double));
	printf("CUDA malloc d_part; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part, DBL_MIN, BLOCK_SIZE*sizeof(double));
	printf("CUDA memset d_part to DBL_MIN: %s\n", hipGetErrorString(err));

	// load d_max to device memory
	double *d_max;
	err = hipMalloc(&d_max, sizeof(double));
	printf("CUDA malloc d_max; %s\n", hipGetErrorString(err));
	err = hipMemset(d_max, DBL_MIN, sizeof(double));
	printf("CUDA memset d_max to DBL_MIN: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((A.width*A.height + dimBlock.x - 1)/dimBlock.x);
	//int blockDim_2 = NearestPowerOf2(d_A.width*d_A.height);
	//printf("nearest power of 2 (blockDim_2): %d\n",blockDim_2);
	// first pass
	maxReduceKernel<<<dimGrid, dimBlock>>>(d_A.elements, d_A.width*d_A.height, d_part);
	err = hipDeviceSynchronize();
	printf("Run kernel 1st pass: %s\n", hipGetErrorString(err));
	// second pass
	dimGrid = dim3(1);
	maxReduceKernel<<<dimGrid, dimBlock>>>(d_part, BLOCK_SIZE, d_max);
	err = hipDeviceSynchronize();
	printf("Run kernel 2nd pass: %s\n", hipGetErrorString(err));

	// read max from device memory
	double max;
	err = hipMemcpy(&max, d_max, sizeof(double), hipMemcpyDeviceToHost);
	printf("Copy max off of device: %s\n",hipGetErrorString(err));
	
	// stop the timer
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("Time elapsed: %f ms\n", time);



	// free device memory
	hipFree(d_A.elements);
	hipFree(d_max);
	return max;
}

// matrix zeros kernel called by zeros()
__global__
void reshapeKernel(Matrix d_In, Matrix d_Out) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(y > d_In.height || x > d_In.width) return;
	int c = x * d_In.height + y;
	d_Out.elements[(c%d_Out.height)*d_Out.width+(c/d_Out.height)] = d_In.elements[(c%d_In.height)*d_In.width+(c/d_In.height)];
}

void reshape(Matrix In, Matrix Out) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	reshapeKernel<<<dimGrid, dimBlock>>>(d_In, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

__global__
void maxReduceKernel(double *elements, int size, double *d_part) {
	// Reduction max, works for any blockDim.x:
	int  thread2;
	double temp;
	__shared__ double sdata[BLOCK_SIZE];
	
	// Load max from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		sdata[threadIdx.x] = elements[idx];
	else
		sdata[threadIdx.x] = DBL_MIN;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE;	// Total number of threads, rounded up to the next power of two
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;

			// Skipping the fictious threads blockDim.x ... blockDim_2-1
			if (thread2 < blockDim.x) {
				// Get the shared value stored by another thread 
				temp = sdata[thread2];
				if (temp > sdata[threadIdx.x]) 
					 sdata[threadIdx.x] = temp;
			}
		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the max to d_max
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

double maxOfArray(double* A, int elements) {
	hipEvent_t start, stop;
	float time;
	// create events and start the timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	// load A to device memory
	double* d_A;
	size_t size = elements * sizeof(double);
	hipError_t err = hipMalloc(&d_A, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load d_part to device memory
	double *d_part;
	err = hipMalloc(&d_part, BLOCK_SIZE*sizeof(double));
	printf("CUDA malloc d_part; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part, DBL_MIN, BLOCK_SIZE*sizeof(double));
	printf("CUDA memset d_part to DBL_MIN: %s\n", hipGetErrorString(err));

	// load d_max to device memory
	double *d_max;
	err = hipMalloc(&d_max, sizeof(double));
	printf("CUDA malloc d_max; %s\n", hipGetErrorString(err));
	err = hipMemset(d_max, DBL_MIN, sizeof(double));
	printf("CUDA memset d_max to DBL_MIN: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((elements + dimBlock.x - 1)/dimBlock.x);
	//int blockDim_2 = NearestPowerOf2(d_A.width*d_A.height);
	//printf("nearest power of 2 (blockDim_2): %d\n",blockDim_2);
	// first pass
	maxReduceKernel<<<dimGrid, dimBlock>>>(d_A, elements, d_part);
	err = hipDeviceSynchronize();
	printf("Run kernel 1st pass: %s\n", hipGetErrorString(err));
	// second pass
	dimGrid = dim3(1);
	maxReduceKernel<<<dimGrid, dimBlock>>>(d_part, BLOCK_SIZE, d_max);
	err = hipDeviceSynchronize();
	printf("Run kernel 2nd pass: %s\n", hipGetErrorString(err));

	// read max from device memory
	double max;
	err = hipMemcpy(&max, d_max, sizeof(double), hipMemcpyDeviceToHost);
	printf("Copy max off of device: %s\n",hipGetErrorString(err));
	
	// stop the timer
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("Time elapsed: %f ms\n", time);

	// free device memory
	hipFree(d_A);
	hipFree(d_max);
	return max;
}
