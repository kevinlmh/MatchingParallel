#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>

#define BLOCK_SIZE 32 

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
  int width;
  int height;
	double* elements;
} Matrix;

//function to print a matrix
void printMatrix(Matrix A) {
	printf("\n");
	for (int i=0; i<A.height; i++) {
		for (int j=0; j<A.width; j++) {
			printf("%.4f ", A.elements[i*A.width+j]); 
		}
		printf("\n");
	}
	printf("\n");
}

// matrix zeros kernel called by zeros()
__global__
void zerosKernel(Matrix d_A) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	d_A.elements[row*d_A.width+col] = 0;
}

void zeros(Matrix A) {
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	zerosKernel<<<dimGrid, dimBlock>>>(d_A);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
}

//matrix transpose kernel called by transpose()
__global__
void transposeKernel(Matrix d_A, Matrix d_B){
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_B.elements[col*d_B.width + row] = d_A.elements[row*d_A.width + col];
}

void transpose(Matrix A, Matrix B){

// load A and B to device memory
	Matrix d_A;
	Matrix d_B;
	d_A.width = A.width;
	d_B.width = B.width;
	d_A.height = A.height;
	d_B.height = B.height;
	size_t size = A.width * A.height * sizeof(double);

	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	hipError_t errB = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(errB));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(errB));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	transposeKernel<<<dimGrid, dimBlock>>>(d_A, d_B);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n",hipGetErrorString(err));
	errB = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy off of device: %s\n", hipGetErrorString(errB));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
}

// matrix ones kernel called by ones()
__global__
void onesKernel(Matrix d_A) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	d_A.elements[row*d_A.width+col] = 1;
}

void ones(Matrix A) {
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	zerosKernel<<<dimGrid, dimBlock>>>(d_A);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
}

// check if a square matrix is symmetric
__global__
void isSymmetricKernel(Matrix d_A, Matrix d_B){

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	if(d_A.elements[row*d_A.width+col] != d_A.elements[row + col*d_A.width])
		d_results = 0;
}

int isSymmetric(Matrix A, int *result) {
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load result to device memory
	int result = 1;
	int *d_result;
	err = hipMalloc(&d_result, sizeof(int));
	printf("CUDA malloc d_result: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);	
	printf("Copy result to device: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	isSymmetricKernel<<<dimGrid, dimBlock>>>(d_A, d_result);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	//read result from fdevice memory
	err = hipMemcpy(&result, d_result, size, hipMemcpyDeviceToHost);
	printf("Copy result off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_result);

	return result;
}

// check if a matrix is symmetric
__global__
void isSymmetricEpsKernel(Matrix d_A, int *result, double eps){

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	if(d_A.elements[row*d_A.width+col] + eps < d_A.elements[row + col*d_A.width] && 
		d_A.elements[row*d_A.width+col] - eps > d_A.elements[row + col*d_A.width])
	
		d_result = 0;
}

int isSymmetricEps(Matrix A, int *result, double eps) {
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load result to device memory
	int result = 1;
	int *result;
	err = hipMalloc(&d_results, sizeof(int));
	printf("CUDA malloc d_result: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);	
	printf("Copy result to device: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	isSymmetricEpsKernel<<<dimGrid, dimBlock>>>(d_A, d_result, eps);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read result from device memory
	err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
	printf("Copy result off of device: %s\n",hipGetErrorString(errA));


	// free device memory
	hipFree(d_A.elements);
	hipFree(d_result);
	
	return result;

}

//create an m-by-n tiling of a given matrix
__global__
void repmatKernel(Matrix d_A, Matrix d_B){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	int h_reps = d_B.width / d_A.width;
	int v_reps = d_B.height / d_A.height;
	for(int i=0; i < h_reps; i++){
		for(int j=0; j < v_reps; j++){
			d_B.elements[row*d_B.width + col + d_A.width*i + d_B.width*j*d_A.height] = d_A.elements[row*d_A.width + col];
		}
	}
}

void repmat(Matrix A, Matrix B){
// load A and B to device memory
	Matrix d_A;
	Matrix d_B;
	d_A.width = A.width;
	d_B.width = B.width;
	d_A.height = A.height;
	d_B.height = B.height;
	size_t sizeA = A.width * A.height * sizeof(double);
	size_t sizeB = B.width * B.height * sizeof(double);

	hipError_t err = hipMalloc(&d_A.elements, sizeA);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	hipMemcpy(d_A.elements, A.elements, sizeA, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	hipError_t errB = hipMalloc(&d_B.elements, sizeB);
	printf("CUDA malloc B: %s\n", hipGetErrorString(errB));
	hipMemcpy(d_B.elements, B.elements, sizeB, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(errB));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	repmatKernel<<<dimGrid, dimBlock>>>(d_A, d_B);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, sizeA, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(err));
	err = hipMemcpy(B.elements, d_B.elements, sizeB, hipMemcpyDeviceToHost);
	printf("Copy B off of device: %s\n", hipGetErrorString(errB));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
}

__global__
void matSubKernel(Matrix d_A, Matrix d_B, Matrix d_C){
int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	d_C.elements[row*d_A.width + col] = d_A.elements[row*d_A.width + col] - d_B.elements[row*d_A.width + col];
}

void matSub(Matrix A, Matrix B, Matrix C){

// load A, B, and C to device memory
	Matrix d_A;
	Matrix d_B;
	Matrix d_C;
	d_A.width = A.width;
	d_B.width = B.width;
	d_C.width = C.width;
	d_A.height = A.height;
	d_B.height = B.height;
	d_C.height = C.height;
	size_t size = A.width * A.height * sizeof(double);

	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(err));

	err = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));
	hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);
	printf("Copy C to device: %s\n", hipGetErrorString(err));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matSubKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(err));
	err = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy B off of device: %s\n", hipGetErrorString(err));
	err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n", hipGetErrorString(err));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

__global__
void matAddKernel(Matrix d_A, Matrix d_B, Matrix d_C){

	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_C.elements[row*d_C.width + col] = d_A.elements[row*d_A.width + col] + d_B.elements[row*d_B.width + col];
}

void matAdd(Matrix A, Matrix B, Matrix C){

// load A, B, and C to device memory
	Matrix d_A;
	Matrix d_B;
	Matrix d_C;
	d_A.width = A.width;
	d_B.width = B.width;
	d_C.width = C.width;
	d_A.height = A.height;
	d_B.height = B.height;
	d_C.height = C.height;
	size_t size = A.width * A.height * sizeof(double);

	hipError_t errA = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(errA));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(errA));

	hipError_t errB = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(errB));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(errB));

	hipError_t errC = hipMalloc(&d_C.elements, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(errC));
	hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);
	printf("Copy C to device: %s\n", hipGetErrorString(errC));

// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matAddKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipError_t err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	printf("Copy A off of device: %s\n",hipGetErrorString(errA));
	err = hipMemcpy(B.elements, d_B.elements, size, hipMemcpyDeviceToHost);
	printf("Copy B off of device: %s\n", hipGetErrorString(errB));
	err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	printf("Copy C off of device: %s\n", hipGetErrorString(errC));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}


// matrix matTimesScaler kernel called by matTimesScaler()
__global__
void matTimesScalerKernel(Matrix d_In, double scaler, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	int idx = row * d_In.width +  col;
	d_Out.elements[idx] = d_In.elements[idx] * scaler;
}

void matTimesScaler(Matrix In, double scaler, Matrix Out) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	matTimesScalerKernel<<<dimGrid, dimBlock>>>(d_In, scaler, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// matrix matPlusScaler kernel called by matPlusScaler()
__global__
void matPlusScalerKernel(Matrix d_In, double scaler, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	int idx = row * d_In.width +  col;
	d_Out.elements[idx] = d_In.elements[idx] + scaler;
}

void matPlusScaler(Matrix In, double scaler, Matrix Out) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	matPlusScalerKernel<<<dimGrid, dimBlock>>>(d_In, scaler, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);

}

// matrix matDiv kernel called by matDiv()
__global__
void matDivKernel(Matrix d_A, Matrix d_B, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row*d_A.width+col;
	if(row > d_A.height || col > d_A.width) return;
	d_Out.elements[idx] = d_A.elements[idx] / d_B.elements[idx];
}

void matDiv(Matrix A, Matrix B, Matrix Out) {
	if (A.width != B.width || A.height != B.height) {
		printf("Input matrices must have the same dimension!\n");
		return;
	}
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix A to device: %s\n", hipGetErrorString(err));
	
	// load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	err = hipMalloc(&d_B.elements, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix B to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matDivKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_Out.elements);

}


// matrix getCol kernel called by getCol()
__global__
void getColKernel(Matrix d_In, Matrix d_Out, int num) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	if(col == num) 
		d_Out.elements[row] = d_In.elements[row*d_In.width+col];
}

void getCol(Matrix In, Matrix Out, int num) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	getColKernel<<<dimGrid, dimBlock>>>(d_In, d_Out, num);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output row off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// matrix getRow kernel called by getRow()
__global__
void getRowKernel(Matrix d_In, Matrix d_Out, int num) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	if(row == num) 
		d_Out.elements[col] = d_In.elements[row*d_In.width+col];
}

void getRow(Matrix In, Matrix Out, int num) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	getRowKernel<<<dimGrid, dimBlock>>>(d_In, d_Out, num);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output row off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// matrix indexOfElement kernel called by indexOfElement()
__global__
void indexOfElementKernel(Matrix d_A, double element, int *index) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	int idx = row*d_A.width+col;
	if (d_A.elements[idx] == element)
		*(index) = idx;
}

int indexOfElement(Matrix A, double element) {
	int index;	
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load index to device memory
	int *d_index;
	hipMemset(d_index, -1, sizeof(int));
	err = hipMalloc(&d_index, sizeof(int));
	printf("CUDA malloc index; %s\n", hipGetErrorString(err));
	hipMemcpy(d_index, &index, sizeof(int), hipMemcpyHostToDevice);
	printf("Copy index to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	indexOfElementKernel<<<dimGrid, dimBlock>>>(d_A, element, d_index);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read index from device memory
	err = hipMemcpy(&index, d_index, sizeof(int), hipMemcpyDeviceToHost);
	printf("Copy index off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_index);
	return index;
}

__global__
void maxReduceKernel(double *elements, int size, double *d_part) {
	// Reduction max, works for any blockDim.x:
	int  thread2;
	double temp;
	__shared__ double sdata[BLOCK_SIZE];
	
	// Load max from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		sdata[threadIdx.x] = elements[idx];
	else
		sdata[threadIdx.x] = DBL_MIN;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE;	// Total number of threads, rounded up to the next power of two
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;

			// Skipping the fictious threads blockDim.x ... blockDim_2-1
			if (thread2 < blockDim.x) {
				// Get the shared value stored by another thread 
				temp = sdata[thread2];
				if (temp > sdata[threadIdx.x]) 
					 sdata[threadIdx.x] = temp;
			}
		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the max to d_max
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

double maxOfMatrix(Matrix A) {
	hipEvent_t start, stop;
	float time;
	// create events and start the timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load d_part to device memory
	double *d_part;
	err = hipMalloc(&d_part, BLOCK_SIZE*sizeof(double));
	printf("CUDA malloc d_part; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part, DBL_MIN, BLOCK_SIZE*sizeof(double));
	printf("CUDA memset d_part to DBL_MIN: %s\n", hipGetErrorString(err));

	// load d_max to device memory
	double *d_max;
	err = hipMalloc(&d_max, sizeof(double));
	printf("CUDA malloc d_max; %s\n", hipGetErrorString(err));
	err = hipMemset(d_max, DBL_MIN, sizeof(double));
	printf("CUDA memset d_max to DBL_MIN: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((A.width*A.height + dimBlock.x - 1)/dimBlock.x);

	// first pass
	maxReduceKernel<<<dimGrid, dimBlock>>>(d_A.elements, d_A.width*d_A.height, d_part);
	err = hipDeviceSynchronize();
	printf("Run kernel 1st pass: %s\n", hipGetErrorString(err));
	// second pass
	dimGrid = dim3(1);
	maxReduceKernel<<<dimGrid, dimBlock>>>(d_part, BLOCK_SIZE, d_max);
	err = hipDeviceSynchronize();
	printf("Run kernel 2nd pass: %s\n", hipGetErrorString(err));

	// read max from device memory
	double max;
	err = hipMemcpy(&max, d_max, sizeof(double), hipMemcpyDeviceToHost);
	printf("Copy max off of device: %s\n",hipGetErrorString(err));
	
	// stop the timer
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("Time elapsed: %f ms\n", time);

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_max);
	return max;
}

// matrix reshape kernel called by reshape()
__global__
void reshapeKernel(Matrix d_In, Matrix d_Out) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(y > d_In.height || x > d_In.width) return;
	int c = x * d_In.height + y;
	d_Out.elements[(c%d_Out.height)*d_Out.width+(c/d_Out.height)] = d_In.elements[(c%d_In.height)*d_In.width+(c/d_In.height)];
}

void reshape(Matrix In, Matrix Out) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
	d_Out.width = Out.width; d_Out.height = Out.height;
	size = Out.width * Out.height * sizeof(double);
	hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	reshapeKernel<<<dimGrid, dimBlock>>>(d_In, d_Out);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

__global__
void minReduceKernel(double *elements, int size, double *d_part) {
	// Reduction min, works for any blockDim.x:
	int  thread2;
	double temp;
	__shared__ double sdata[BLOCK_SIZE];
	
	// Load min from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		sdata[threadIdx.x] = elements[idx];
	else
		sdata[threadIdx.x] = DBL_MAX;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE;	// Total number of threads, rounded up to the next power of two
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;

			// Skipping the fictious threads blockDim.x ... blockDim_2-1
			if (thread2 < blockDim.x) {
				// Get the shared value stored by another thread 
				temp = sdata[thread2];
				if (temp < sdata[threadIdx.x]) 
					 sdata[threadIdx.x] = temp;
			}
		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the min to d_min
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

double minOfMatrix(Matrix A) {
	hipEvent_t start, stop;
	float time;
	// create events and start the timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load d_part to device memory
	double *d_part;
	err = hipMalloc(&d_part, BLOCK_SIZE*sizeof(double));
	printf("CUDA malloc d_part; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part, DBL_MAX, BLOCK_SIZE*sizeof(double));
	printf("CUDA memset d_part to DBL_MAX: %s\n", hipGetErrorString(err));

	// load d_min to device memory
	double *d_min;
	err = hipMalloc(&d_min, sizeof(double));
	printf("CUDA malloc d_min; %s\n", hipGetErrorString(err));
	err = hipMemset(d_min, DBL_MAX, sizeof(double));
	printf("CUDA memset d_min to DBL_MAX: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((A.width*A.height + dimBlock.x - 1)/dimBlock.x);

	// first pass
	minReduceKernel<<<dimGrid, dimBlock>>>(d_A.elements, d_A.width*d_A.height, d_part);
	err = hipDeviceSynchronize();
	printf("Run kernel 1st pass: %s\n", hipGetErrorString(err));
	// second pass
	dimGrid = dim3(1);
	minReduceKernel<<<dimGrid, dimBlock>>>(d_part, BLOCK_SIZE, d_min);
	err = hipDeviceSynchronize();
	printf("Run kernel 2nd pass: %s\n", hipGetErrorString(err));

	// read min from device memory
	double min;
	err = hipMemcpy(&min, d_min, sizeof(double), hipMemcpyDeviceToHost);
	printf("Copy min off of device: %s\n",hipGetErrorString(err));
	
	// stop the timer
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("Time elapsed: %f ms\n", time);

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_min);
	return min;
}

__global__
void minArrayKernel(double *elements, int size, double *d_part) {
	// Reduction min, works for any blockDim.x:
	int  thread2;
	double temp;
	__shared__ double sdata[BLOCK_SIZE];
	
	// Load min from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		sdata[threadIdx.x] = elements[idx];
	else
		sdata[threadIdx.x] = DBL_MAX;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE;	// Total number of threads, rounded up to the next power of two
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;

			// Skipping the fictious threads blockDim.x ... blockDim_2-1
			if (thread2 < blockDim.x) {
				// Get the shared value stored by another thread 
				temp = sdata[thread2];
				if (temp < sdata[threadIdx.x]) 
					 sdata[threadIdx.x] = temp;
			}
		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the min to d_min
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

double minOfArray(double* A, int elements) {
	hipEvent_t start, stop;
	float time;
	// create events and start the timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	// load A to device memory
	double* d_A;
	size_t size = elements * sizeof(double);
	hipError_t err = hipMalloc(&d_A, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load d_part to device memory
	double *d_part;
	err = hipMalloc(&d_part, BLOCK_SIZE*sizeof(double));
	printf("CUDA malloc d_part; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part, DBL_MAX, BLOCK_SIZE*sizeof(double));
	printf("CUDA memset d_part to DBL_MAX: %s\n", hipGetErrorString(err));

	// load d_min to device memory
	double *d_min;
	err = hipMalloc(&d_min, sizeof(double));
	printf("CUDA malloc d_min; %s\n", hipGetErrorString(err));
	err = hipMemset(d_min, DBL_MAX, sizeof(double));
	printf("CUDA memset d_min to DBL_MAX: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((elements + dimBlock.x - 1)/dimBlock.x);
	
	// first pass
	minArrayKernel<<<dimGrid, dimBlock>>>(d_A, elements, d_part);
	err = hipDeviceSynchronize();
	printf("Run kernel 1st pass: %s\n", hipGetErrorString(err));

	// second pass
	dimGrid = dim3(1);
	minArrayKernel<<<dimGrid, dimBlock>>>(d_part, BLOCK_SIZE, d_min);
	err = hipDeviceSynchronize();
	printf("Run kernel 2nd pass: %s\n", hipGetErrorString(err));

	// read min from device memory
	double min;
	err = hipMemcpy(&min, d_min, sizeof(double), hipMemcpyDeviceToHost);
	printf("Copy min off of device: %s\n",hipGetErrorString(err));
	
	// stop the timer
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("Time elapsed: %f ms\n", time);

	// free device memory
	hipFree(d_A);
	hipFree(d_min);
	return min;
}

__global__
void arraySumKernel(double *elements, int size, double *d_part) {
	// Reduction sum, works for any blockDim.x:
	int  thread2;
	double temp;
	__shared__ double sdata[BLOCK_SIZE];
	
	// Load sum from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		sdata[threadIdx.x] = elements[idx];
	else
		sdata[threadIdx.x] = 0;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE;	// Total number of threads, rounded up to the next power of two
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;

			// Skipping the fictious threads blockDim.x ... blockDim_2-1
			if (thread2 < blockDim.x) {
				// Get the shared value stored by another thread 
				temp = sdata[thread2];
					 sdata[threadIdx.x] += temp;
			}
		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the sum to d_sum
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

double arraySum(double* A, int elements) {
	hipEvent_t start, stop;
	float time;
	// create events and start the timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	// load A to device memory
	double* d_A;
	size_t size = elements * sizeof(double);
	hipError_t err = hipMalloc(&d_A, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load d_part to device memory
	double *d_part;
	err = hipMalloc(&d_part, BLOCK_SIZE*sizeof(double));
	printf("CUDA malloc d_part; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part, 0, BLOCK_SIZE*sizeof(double));
	printf("CUDA memset d_part to 0: %s\n", hipGetErrorString(err));

	// load d_sum to device memory
	double *d_sum;
	err = hipMalloc(&d_sum, sizeof(double));
	printf("CUDA malloc d_sum; %s\n", hipGetErrorString(err));
	err = hipMemset(d_sum, 0, sizeof(double));
	printf("CUDA memset d_sum to 0: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((elements + dimBlock.x - 1)/dimBlock.x);

	// first pass
	arraySumKernel<<<dimGrid, dimBlock>>>(d_A, elements, d_part);
	err = hipDeviceSynchronize();
	printf("Run kernel 1st pass: %s\n", hipGetErrorString(err));
	// second pass
	dimGrid = dim3(1);
	arraySumKernel<<<dimGrid, dimBlock>>>(d_part, BLOCK_SIZE, d_sum);
	err = hipDeviceSynchronize();
	printf("Run kernel 2nd pass: %s\n", hipGetErrorString(err));

	// read sum from device memory
	double sum;
	err = hipMemcpy(&sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
	printf("Copy sum off of device: %s\n",hipGetErrorString(err));
	
	// stop the timer
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("Time elapsed: %f ms\n", time);

	// free device memory
	hipFree(d_A);
	hipFree(d_sum);
	return sum;
}

__global__
void matSumKernel(double *elements, int size, double *d_part) {
	// Reduction sum, works for any blockDim.x:
	int  thread2;
	double temp;
	__shared__ double sdata[BLOCK_SIZE];
	
	// Load sum from global memory
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		sdata[threadIdx.x] = elements[idx];
	else
		sdata[threadIdx.x] = 0;
	
	// Synchronize to make sure data is loaded before starting the comparison
  __syncthreads();

	int nTotalThreads = BLOCK_SIZE;	// Total number of threads, rounded up to the next power of two
	 
	while(nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
	 
		if (threadIdx.x < halfPoint) {
			thread2 = threadIdx.x + halfPoint;

			// Skipping the fictious threads blockDim.x ... blockDim_2-1
			if (thread2 < blockDim.x) {
				// Get the shared value stored by another thread 
				temp = sdata[thread2];
				if (temp > sdata[threadIdx.x]) 
					 sdata[threadIdx.x] += temp;
			}
		}
		__syncthreads();
	 
		// Reducing the binary tree size by two:
		nTotalThreads = halfPoint;
	}
	
	// thread 0 copy the sum to d_sum
	if (threadIdx.x == 0) {
		d_part[blockIdx.x] = sdata[threadIdx.x];
	}
}

double matSum(Matrix A) {
	hipEvent_t start, stop;
	float time;
	// create events and start the timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load d_part to device memory
	double *d_part;
	err = hipMalloc(&d_part, BLOCK_SIZE*sizeof(double));
	printf("CUDA malloc d_part; %s\n", hipGetErrorString(err));
	err = hipMemset(d_part, 0, BLOCK_SIZE*sizeof(double));
	printf("CUDA memset d_part to 0 %s\n", hipGetErrorString(err));

	// load d_sum to device memory
	double *d_sum;
	err = hipMalloc(&d_sum, sizeof(double));
	printf("CUDA malloc d_sum; %s\n", hipGetErrorString(err));
	err = hipMemset(d_sum, 0, sizeof(double));
	printf("CUDA memset d_sum to 0: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((A.width*A.height + dimBlock.x - 1)/dimBlock.x);

	// first pass
	matSumKernel<<<dimGrid, dimBlock>>>(d_A.elements, d_A.width*d_A.height, d_part);
	err = hipDeviceSynchronize();
	printf("Run kernel 1st pass: %s\n", hipGetErrorString(err));
	// second pass
	dimGrid = dim3(1);
	matSumKernel<<<dimGrid, dimBlock>>>(d_part, BLOCK_SIZE, d_sum);
	err = hipDeviceSynchronize();
	printf("Run kernel 2nd pass: %s\n", hipGetErrorString(err));

	// read sum from device memory
	double sum;
	err = hipMemcpy(&sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
	printf("Copy sum off of device: %s\n",hipGetErrorString(err));
	
	// stop the timer
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("Time elapsed: %f ms\n", time);

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_sum);
	return sum;
}
